#include "hip/hip_runtime.h"
#include "./FzbGetIllumination.cuh"
#include "FzbCollisionDetection.cuh"

__device__ float DistributionGGX(const glm::vec3& N, const glm::vec3& H, float roughness)
{
	float a = roughness * roughness;
	float a2 = a * a;
	float NdotH = max(glm::dot(N, H), 0.0);
	float NdotH2 = NdotH * NdotH;

	float nom = a2;
	float denom = (NdotH2 * (a2 - 1.0) + 1.0);
	denom = PI * denom * denom;

	return nom / denom;
}
__device__ float GeometrySchlickGGX(float NdotV, float roughness)
{
	float r = (roughness + 1.0);
	float k = (r * r) / 8.0;

	float nom = NdotV;
	float denom = NdotV * (1.0 - k) + k;

	return nom / denom;
}

__device__ float GeometrySmith(const glm::vec3& N, const glm::vec3& V, const glm::vec3& L, float roughness)
{
	float NdotV = max(glm::dot(N, V), 0.0);
	float NdotL = max(glm::dot(N, L), 0.0);
	float ggx2 = GeometrySchlickGGX(NdotV, roughness);
	float ggx1 = GeometrySchlickGGX(NdotL, roughness);

	return ggx1 * ggx2;
}

__device__ glm::vec3 fresnelSchlick(float cosTheta, const glm::vec3& F0)
{
	return F0 + (1.0f - F0) * pow(glm::clamp(1.0f - cosTheta, 0.0f, 1.0f), 5.0f);
}

__device__ glm::vec3 getBSDF(const FzbTriangleAttribute& triangleAttribute, const glm::vec3& incidence, const glm::vec3& outgoing) {
	if(triangleAttribute.materialType == 0) return glm::vec3(PI_countdown) * triangleAttribute.albedo;
	else if (triangleAttribute.materialType == 1) {
		glm::vec3 h = normalize(incidence + outgoing);
		float NDF = DistributionGGX(triangleAttribute.normal, h, triangleAttribute.roughness);
		float G = GeometrySmith(triangleAttribute.normal, outgoing, incidence, triangleAttribute.roughness);
		glm::vec3 F = fresnelSchlick(glm::max(glm::dot(h, outgoing), 0.0f), triangleAttribute.albedo);

		glm::vec3 fr = NDF * G * F;
		float denominator = 4.0f * glm::max(glm::dot(triangleAttribute.normal, outgoing), 0.0f) * glm::max(glm::dot(triangleAttribute.normal, incidence), 0.0f) + 0.01f;
		return fr /= denominator;
	}
	return glm::vec3(0.0f);
}

/*
���شӹ�Դ����һ��ײ�����radiance
����Ӧ�����Ż������������й�Դ���в�������������ķ�������Ŀǰ�ȷ���
*/
__device__ glm::vec3 NEE(FzbTriangleAttribute& triangleAttribute, FzbRay& ray, const FzbRayTracingLightSet* lightSet,
	const float* __restrict__ vertices, const hipTextureObject_t* __restrict__ materialTextures,
	const FzbBvhNode* __restrict__ bvhNodeArray, const FzbBvhNodeTriangleInfo* __restrict__ bvhTriangleInfoArray, uint32_t& randomNumberSeed) {
	glm::vec3 radiance = glm::vec3(0.0f);
	FzbRay tempRay;
	FzbTriangleAttribute hitTriangleAttribute;
	for (int i = 0; i < lightSet->pointLightCount; ++i) {
		const FzbRayTracingPointLight& light = lightSet->pointLightInfoArray[i];
		glm::vec3 direction = light.worldPos - ray.hitPos;
		tempRay.depth = FLT_MAX;
		tempRay.direction = glm::normalize(direction);
		bool hit = sceneCollisionDetection(bvhNodeArray, bvhTriangleInfoArray, vertices, materialTextures, tempRay, hitTriangleAttribute, false);
		if (!hit) continue;
		else if (abs(tempRay.depth - glm::length(direction)) > 0.1f) continue;
		float r2 = glm::length(direction); r2 *= r2;
		float cosTheta = glm::clamp(glm::dot(triangleAttribute.normal, tempRay.direction), 0.0f, 1.0f);
		radiance += cosTheta * light.radiantIntensity / r2 * getBSDF(triangleAttribute, tempRay.direction, -ray.direction);
	}
	for (int i = 0; i < lightSet->areaLightCount; ++i) {
		const FzbRayTracingAreaLight& light = lightSet->areaLightInfoArray[i];
		float randomNumberX = rand(randomNumberSeed);
		float randomNumberY = rand(randomNumberSeed);
		glm::vec3 lightPos = glm::vec3(light.worldPos + randomNumberX * light.edge0 + randomNumberY * light.edge1);
		glm::vec3 direction = lightPos - ray.hitPos;
		tempRay.startPos = ray.hitPos + direction * 0.001f;
		tempRay.depth = FLT_MAX;
		tempRay.direction = glm::normalize(direction);
		float r = glm::length(direction);

		bool hit = sceneCollisionDetection(bvhNodeArray, bvhTriangleInfoArray, vertices, materialTextures, tempRay, hitTriangleAttribute, false);
		if (!hit) continue;
		else if (abs(tempRay.depth - r) > 0.1f) continue;
		glm::vec3 lightRadiance_cosTheta = light.radiance * glm::clamp(glm::dot(triangleAttribute.normal, tempRay.direction), 0.0f, 1.0f);
		lightRadiance_cosTheta *= getBSDF(triangleAttribute, tempRay.direction, -ray.direction) * light.area;	//bsdf / pdf
		lightRadiance_cosTheta *= glm::clamp(glm::dot(-light.normal, tempRay.direction), 0.0f, 1.0f);	//΢�ֵ�λ��dw��ΪdA
		r = glm::max(r, 1.0f);
		lightRadiance_cosTheta /= r * r;
		radiance += lightRadiance_cosTheta;
	}
	return radiance;
}

__device__ glm::vec3 getRadiance(FzbTriangleAttribute& triangleAttribute, FzbRay& ray, const FzbRayTracingLightSet* lightSet,
	const float* __restrict__ vertices, const hipTextureObject_t* __restrict__ materialTextures,
	const FzbBvhNode* __restrict__ bvhNodeArray, const FzbBvhNodeTriangleInfo* __restrict__ bvhTriangleInfoArray, uint32_t& randomNumberSeed) {
	glm::vec3 radiance = glm::vec3(0.0f);
	radiance += NEE(triangleAttribute, ray, lightSet, vertices, materialTextures, bvhNodeArray, bvhTriangleInfoArray, randomNumberSeed);
	radiance += triangleAttribute.emissive;
	return radiance;
}

