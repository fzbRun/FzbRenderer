#include "hip/hip_runtime.h"
#include "./FzbRayTracingInitSource.cuh"
#include "../../common/FzbRenderer.h"

__constant__ FzbPathTracingCameraInfo systemCameraInfo;
__constant__ FzbRayTracingMaterialUniformObject materialInfoArray[maxMaterialCount];
__constant__ bool useCudaRandom;
__constant__ hiprandState* systemRandomNumberStates;
__constant__ uint32_t systemRandomNumberSeed;

__constant__ uint32_t systemPointLightCount;
__constant__ FzbRayTracingPointLight systemPointLightInfoArray[maxPointLightCount];
__constant__ uint32_t systemAreaLightCount;
__constant__ FzbRayTracingAreaLight systemAreaLightInfoArray[maxAreaLightCount];

FzbRayTracingSourceManager_Cuda::FzbRayTracingSourceManager_Cuda() {};
void FzbRayTracingSourceManager_Cuda::initRayTracingSource(FzbRayTracingCudaSourceSet& sourceSet) {
	if (getCudaDeviceForVulkanPhysicalDevice(FzbRenderer::globalData.physicalDevice) == hipInvalidDeviceId) {
		throw std::runtime_error("CUDA��Vulkan�õĲ���ͬһ��GPU������");
	}

	//��������buffer
	resultBufferExtMem = importVulkanMemoryObjectFromNTHandle(sourceSet.rayTracingResultBuffer.handle, sourceSet.rayTracingResultBuffer.size, false);
	resultBuffer = (float4*)mapBufferOntoExternalMemory(resultBufferExtMem, 0, sourceSet.rayTracingResultBuffer.size);
	extRayTracingFinishedSemaphore = importVulkanSemaphoreObjectFromNTHandle(sourceSet.rayTracingFinishedSemphore.handle);

	//��ȡ������������
	vertexExtMem = importVulkanMemoryObjectFromNTHandle(sourceSet.sceneVertices.handle, sourceSet.sceneVertices.size, false);
	vertices = (float*)mapBufferOntoExternalMemory(vertexExtMem, 0, sourceSet.sceneVertices.size);

	//��ȡ����Material��Ϣ
	uint32_t textureCount = sourceSet.sceneTextures.size();
	this->textureExtMems.resize(textureCount);
	this->textureMipmap.resize(textureCount);
	this->textureObjects.resize(textureCount);
	uint32_t textureIndex = 0;
	for (auto& sceneTexture : sourceSet.sceneTextures) {
		FzbImage& texture = sceneTexture;
		uint32_t textureSize = texture.width * texture.height * texture.depth * sizeof(uint32_t);
		fromVulkanImageToCudaTexture(FzbRenderer::globalData.physicalDevice, texture, texture.handle, textureSize, false, textureExtMems[textureIndex], textureMipmap[textureIndex], textureObjects[textureIndex], true);
		++textureIndex;
	}
	CHECK(hipMalloc((void**)&materialTextures, textureCount * sizeof(hipTextureObject_t)));
	hipMemcpy(materialTextures, textureObjects.data(), textureCount * sizeof(hipTextureObject_t), hipMemcpyHostToDevice);

	//��materialInfoArrayBuffer����uniform��
	if (sourceSet.sceneMaterialInfoArray.size() > maxMaterialCount) throw std::runtime_error("material��������maxMaterialCount:" + std::to_string(maxMaterialCount) + "����Ҫ����");
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(materialInfoArray), sourceSet.sceneMaterialInfoArray.data(), sourceSet.sceneMaterialInfoArray.size() * sizeof(FzbRayTracingMaterialUniformObject)));

	//�����Ϣ
	//this->extBvhFinishedSemaphore = importVulkanSemaphoreObjectFromNTHandle(sourceSet.bvhSemaphoreHandle);
	this->bvhNodeArray = sourceSet.bvhNodeArray;
	this->bvhTriangleInfoArray = sourceSet.bvhTriangleInfoArray;

	//������Դ��Ϣ
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(systemPointLightCount), &sourceSet.pointLightCount, sizeof(uint32_t)));
	if (sourceSet.pointLightInfoArray.size() > maxPointLightCount) throw std::runtime_error("���Դ��������maxPointLightCount" + std::to_string(maxPointLightCount) + "����Ҫ����");
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(systemPointLightInfoArray), sourceSet.pointLightInfoArray.data(), sourceSet.pointLightInfoArray.size() * sizeof(FzbRayTracingPointLight)));
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(systemAreaLightCount), &sourceSet.areaLightCount, sizeof(uint32_t)));
	if (sourceSet.areaLightInfoArray.size() > maxAreaLightCount) throw std::runtime_error("���Դ��������maxAreaLightCount" + std::to_string(maxAreaLightCount) + "����Ҫ����");
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(systemAreaLightInfoArray), sourceSet.areaLightInfoArray.data(), sourceSet.areaLightInfoArray.size() * sizeof(FzbRayTracingAreaLight)));

	//CHECK(hipMemcpyToSymbol(HIP_SYMBOL(useCudaRandom), &setting.useCudaRandom, sizeof(bool)));
	//if (setting.useCudaRandom) {
	//	VkExtent2D resolution = FzbRenderer::globalData.getResolution();
	//	uint32_t texelCount = resolution.width * resolution.height;
	//	uint32_t rayCount = texelCount * setting.spp;
	//	uint32_t gridSize = (rayCount + blockSize - 1) / blockSize;
	//	hipMalloc(&systemRandomNumberStates_device, rayCount * sizeof(hiprandState));
	//	init_curand_states << <gridSize, blockSize >> > (systemRandomNumberStates_device, time(0), rayCount);
	//	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(systemRandomNumberStates), &systemRandomNumberStates_device, sizeof(hiprandState*)));
	//}

	//����stream
	CHECK(hipStreamCreate(&stream));
}
void FzbRayTracingSourceManager_Cuda::createRuntimeSource() {
	//ΪcameraInfo��ֵ
	FzbPathTracingCameraInfo cameraInfo_host;
	FzbCamera* camera = FzbRenderer::globalData.camera;
	cameraInfo_host.cameraWorldPos = camera->position;
	cameraInfo_host.inversePVMatrix = glm::inverse(camera->GetProjMatrix() * camera->GetViewMatrix());
	VkExtent2D resolution = FzbRenderer::globalData.getResolution();
	cameraInfo_host.screenWidth = resolution.width;
	cameraInfo_host.screenHeight = resolution.height;
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(systemCameraInfo), &cameraInfo_host, sizeof(FzbPathTracingCameraInfo)));
}
void FzbRayTracingSourceManager_Cuda::clean() {
	CHECK(hipFree(resultBuffer));
	CHECK(hipDestroyExternalMemory(resultBufferExtMem));

	CHECK(hipDestroyExternalSemaphore(extRayTracingFinishedSemaphore));

	CHECK(hipFree(vertices));
	CHECK(hipDestroyExternalMemory(vertexExtMem));

	for (int i = 0; i < textureExtMems.size(); ++i) {
		CHECK(hipDestroyTextureObject(textureObjects[i]));
		CHECK(hipFreeMipmappedArray(textureMipmap[i]));
		CHECK(hipDestroyExternalMemory(textureExtMems[i]));
	}
	CHECK(hipFree(materialTextures));
	CHECK(hipStreamDestroy(stream));
}