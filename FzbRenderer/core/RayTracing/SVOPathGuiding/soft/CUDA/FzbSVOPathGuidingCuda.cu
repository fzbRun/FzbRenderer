#include "hip/hip_runtime.h"
#include "./FzbSVOPathGuidingCuda.cuh"
#include "../../../../common/FzbRenderer.h"
#include "../../../CUDA/FzbRayGenerate.cuh"
#include "../../../CUDA/FzbGetTriangleAttribute.cuh"
#include "../../../CUDA/FzbCollisionDetection.cuh"
#include "../../../CUDA/FzbGetIllumination.cuh"

//----------------------------------------------uniformBuffer--------------------------------------
__constant__ FzbSVOPathGuidingCudaSetting svoPathGuidingSetting;

const uint32_t blockSize = 128;
const uint32_t sharedMemorySPP = 2;
//----------------------------------------------SVOPathGuiding����-------------------------------------
__device__ void generateRay_SVOPathGuiding(
	const FzbTriangleAttribute& hitTriangleAttribute, float& pdf, FzbRay& ray, uint32_t& randomNumberSeed,
	FzbSVOPathGuidingCudaSetting& groupSetting)
{
	//����hitPos�ҵ���ǰnode���ĸ�
	FzbSVONodeData_PG nodeData; nodeData.label = 1;
	glm::vec3 nodeGroupStartPos = groupSetting.voxelGroupStartPos;
	glm::vec3 nodeSize = (float)groupSetting.voxelCount * groupSetting.voxelSize;
	uint32_t layerIndex = 1;
	uint32_t nodeDataIndex = 0;

	for (; layerIndex < groupSetting.maxSVOLayer; ++layerIndex) {
		nodeSize /= 2.0f;
		glm::ivec3 nodeIndexXYZ = glm::ivec3((ray.hitPos - nodeGroupStartPos) / nodeSize);
		nodeDataIndex = nodeIndexXYZ.x + 2 * nodeIndexXYZ.y + 4 * nodeIndexXYZ.z + (nodeData.label - 1) * 8;
		nodeData = groupSetting.SVONodes[layerIndex][nodeDataIndex];
		if (nodeData.indivisible) break;

		nodeGroupStartPos += glm::vec3(nodeIndexXYZ) * nodeSize;
	}
	if (nodeData.label == 0) {		//����ע��������դ��������ɢ�ģ��еļ��ο���û��ע���ȥ�����ǹ�����ɢ�̶ȸ�С�����ܴ���
		generateRay(hitTriangleAttribute, pdf, ray, randomNumberSeed);
		return;
	}

	uint32_t layerNodeSum = (nodeData.label - 1) * groupSetting.SVONodeTotalCount;
	uint32_t nodeWeightStartIndex = layerNodeSum;
	uint32_t targetLayerIndex = 1;
	uint32_t targetNodeDataIndex = 0;
	FzbSVONodeData_PG targetNodeData;
	bool getTargetNode = false;
	int childNodeIndex;
	float selectNodWeightSum = 1.0f;
	for (; targetLayerIndex < groupSetting.maxSVOLayer; ++targetLayerIndex) {
		float randomNumber = rand(randomNumberSeed);
		for (childNodeIndex = 0; childNodeIndex < 8; ++childNodeIndex) {
			float weight = groupSetting.SVONodeWeights[nodeWeightStartIndex + childNodeIndex];
			if (randomNumber <= weight) {
				targetNodeData = groupSetting.SVONodes[targetLayerIndex][targetNodeDataIndex + childNodeIndex];
				selectNodWeightSum *= weight;	//�ҵ���node�ĸ����ܶ�
				if (targetNodeData.indivisible) getTargetNode = true;
				else targetNodeDataIndex = (targetNodeData.label - 1) * 8;	//���ӽڵ��ڸò����ʼ����
				break;
			}
			randomNumber -= weight;
		}
		if (getTargetNode || childNodeIndex == 8) break;
		layerNodeSum += groupSetting.SVOLayerInfos[targetLayerIndex - 1].divisibleNodeCount * 8;
		nodeWeightStartIndex = layerNodeSum + targetNodeDataIndex;
	}

	if (glm::length(targetNodeData.irradiance) == 0 || childNodeIndex == 8) {
		generateRay(hitTriangleAttribute, pdf, ray, randomNumberSeed);
		return;
	}

	float targetDistanceX = targetNodeData.AABB.rightX - targetNodeData.AABB.leftX;
	float targetDistanceY = targetNodeData.AABB.rightY - targetNodeData.AABB.leftY;
	float targetDistanceZ = targetNodeData.AABB.rightZ - targetNodeData.AABB.leftZ;

	glm::vec3 nodeCenterPos = glm::vec3(nodeData.AABB.leftX + nodeData.AABB.rightX, nodeData.AABB.leftY + nodeData.AABB.rightY, nodeData.AABB.leftZ + nodeData.AABB.rightZ) * 0.5f;
	glm::vec3 targetNodeCenterPos = glm::vec3(targetNodeData.AABB.leftX + targetNodeData.AABB.rightX, targetNodeData.AABB.leftY + targetNodeData.AABB.rightY, targetNodeData.AABB.leftZ + targetNodeData.AABB.rightZ) * 0.5f;
	glm::vec3 nodeDirection = targetNodeCenterPos - nodeCenterPos;

	glm::vec3 faceArea;
	faceArea.x = targetDistanceY * targetDistanceZ;
	faceArea.y = targetDistanceX * targetDistanceZ;
	faceArea.z = targetDistanceX * targetDistanceY;
	glm::vec3 faceSelectWeight = glm::normalize(faceArea);
	float faceSelectRandomNumber = rand(randomNumberSeed);
	uint32_t faceIndex = faceSelectRandomNumber <= faceSelectWeight.x ? 0 : faceSelectRandomNumber <= faceSelectWeight.x + faceSelectWeight.y ? 1 : 2;

	float selectFaceArea = 1.0f;
	float selectFacePDF = 1.0f;
	glm::vec3 faceNormal = glm::vec3(0.0f);
	glm::vec3 faceStartPos = glm::vec3(targetNodeData.AABB.leftX, targetNodeData.AABB.leftY, targetNodeData.AABB.leftZ);
	if (faceIndex == 0) {
		faceStartPos.x += nodeDirection.x < 0 ? targetDistanceX : 0.0f;	//�����
		faceNormal.x = nodeDirection.x < 0 ? 1.0f : -1.0f;
		selectFaceArea = faceArea.x;
		selectFacePDF = faceSelectWeight.x;
	}
	else if (faceIndex == 1) {
		faceStartPos.y += nodeDirection.y < 0 ? targetDistanceY : 0.0f;	//���±�
		faceNormal.y = nodeDirection.y < 0 ? 1.0f : -1.0f;
		selectFaceArea = faceArea.y;
		selectFacePDF = faceSelectWeight.y;
	}
	else {
		faceStartPos.z += nodeDirection.z < 0 ? targetDistanceZ : 0.0f;	//�ں��
		faceNormal.z = nodeDirection.z < 0 ? 1.0f : -1.0f;
		selectFaceArea = faceArea.z;
		selectFacePDF = faceSelectWeight.z;
	}

	float randomU = rand(randomNumberSeed);		//��ǰnode��AABB�ϵ������
	float randomV = rand(randomNumberSeed);
	bool useSphericalRectangleSample = true;
	float sphericalRectangleSamplePDF = 1.0f;
	if(useSphericalRectangleSample){
		FzbQuadrilateral quadFace;
		quadFace.worldPos = faceStartPos;
		quadFace.normal = faceNormal;
		if (faceIndex == 0) {
			quadFace.edge0 = targetDistanceZ * glm::vec3(0.0f, 0.0f, 1.0f);
			quadFace.edge1 = targetDistanceY * glm::vec3(0.0f, 1.0f, 0.0f);
		}
		else if (faceIndex == 1) {
			quadFace.edge0 = targetDistanceX * glm::vec3(1.0f, 0.0f, 0.0f);
			quadFace.edge1 = targetDistanceZ * glm::vec3(0.0f, 0.0f, 1.0f);
		}
		else {
			quadFace.edge0 = targetDistanceX * glm::vec3(1.0f, 0.0f, 0.0f);
			quadFace.edge1 = targetDistanceY * glm::vec3(0.0f, 1.0f, 0.0f);
		}

		ray.direction = sphericalRectangleSample(quadFace, ray.hitPos, randomU, randomV, sphericalRectangleSamplePDF);
	}
	else {
		glm::vec3 samplePos = faceStartPos;
		if (faceIndex == 0) {
			samplePos.z += randomU * targetDistanceZ;
			samplePos.y += randomV * targetDistanceY;
		}
		else if (faceIndex == 1) {
			samplePos.x += randomU * targetDistanceX;
			samplePos.z += randomV * targetDistanceZ;
		}
		else {
			samplePos.x += randomU * targetDistanceX;
			samplePos.y += randomV * targetDistanceY;
		}

		ray.direction = samplePos - ray.hitPos;
	}

	if (glm::dot(ray.direction, hitTriangleAttribute.normal) <= 0) generateRay(hitTriangleAttribute, pdf, ray, randomNumberSeed);
	else {
		pdf *= selectNodWeightSum * selectFacePDF;	//���ղ���node��AABB���ܺ�����һЩ��Χ�У���Ҫ����pdf���ֲ�
		pdf *= sphericalRectangleSamplePDF;
		if (!useSphericalRectangleSample) 
			pdf *= glm::dot(-glm::normalize(ray.direction), faceNormal) / glm::max(selectFaceArea * glm::max(glm::length(ray.direction), 0.01f), 0.001f);

		ray.direction = glm::normalize(ray.direction);
		ray.startPos = ray.direction * 0.01f + ray.hitPos;
		ray.depth = FLT_MAX;
	}
}
//-------------------------------------------------------------------------------------------------
template<bool useExternSharedMemory>
__global__ void svoPathGuiding_cuda(
	float4* resultBuffer, const float* __restrict__ vertices, const hipTextureObject_t* __restrict__ materialTextures,
	const FzbBvhNode* __restrict__ bvhNodeArray, const FzbBvhNodeTriangleInfo* __restrict__ bvhTriangleInfoArray, const uint32_t rayCount
) {
	extern __shared__ float3 groupResultRadiance[];	//���spp����sharedMemorySPP����ʹ�ù����ڴ棬 ����ΪblockDim / spp, ���6KB
	__shared__ FzbSVOPathGuidingCudaSetting groupSetting;
	__shared__ uint32_t groupFrameCount;
	__shared__ FzbPathTracingCameraInfo groupCameraInfo;				//216B
	__shared__ uint32_t groupRandomNumberSeed;
	__shared__ FzbRayTracingPointLight groupPointLightInfoArray[maxPointLightCount];	//512B
	__shared__ FzbRayTracingAreaLight grouprAreaLightInfoArray[maxAreaLightCount];		//692B
	__shared__ FzbRayTracingLightSet lightSet;

	__shared__ FzbSVOLayerInfo groupSVOLayerInfos[8];
	__shared__ FzbSVONodeData_PG* groupSVONodesArray[8];

	volatile const uint32_t threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
	if (threadIndex >= rayCount) return;

	if (threadIdx.x < systemPointLightCount) groupPointLightInfoArray[threadIdx.x] = systemPointLightInfoArray[threadIdx.x];
	if (threadIdx.x < systemAreaLightCount) grouprAreaLightInfoArray[threadIdx.x] = systemAreaLightInfoArray[threadIdx.x];
	if (threadIdx.x == 0) {
		groupSetting = svoPathGuidingSetting;
		groupFrameCount = systemFrameCount;
		groupCameraInfo = systemCameraInfo;
		groupRandomNumberSeed = systemRandomNumberSeed;
		lightSet.pointLightCount = systemPointLightCount;
		lightSet.areaLightCount = systemAreaLightCount;
		lightSet.pointLightInfoArray = groupPointLightInfoArray;
		lightSet.areaLightInfoArray = grouprAreaLightInfoArray;
	}
	__syncwarp();
	if (threadIdx.x < groupSetting.maxSVOLayer) {
		groupSVOLayerInfos[threadIdx.x] = groupSetting.SVOLayerInfos[threadIdx.x];
		groupSVONodesArray[threadIdx.x] = groupSetting.SVONodes[threadIdx.x];
	}
	__syncwarp();
	if (threadIdx.x == 0) {		//��ȫ���ڴ��Ϊ�����ڴ�
		groupSetting.SVOLayerInfos = groupSVOLayerInfos;
		groupSetting.SVONodes = groupSVONodesArray;
	}
	__syncthreads();

	volatile const uint32_t& spp = groupSetting.spp;		//�Ĵ��������ã�������localMmeory���ǻ�����ֱ����sharedMemory��
	uint32_t resultIndex = threadIndex / spp;	//���ڵڼ���spp����bufferIndex
	uint32_t groupSppIndex = threadIdx.x / spp;		//���ڵڼ���spp
	uint32_t sppLane = threadIndex % spp;	//������&����Ϊspp���ܲ���2���ݴ�
	if (useExternSharedMemory) {
		if (threadIdx.x < blockDim.x / spp) groupResultRadiance[threadIdx.x] = make_float3(0.0f);
	}
	//if (threadIndex < systemCameraInfo.screenWidth * systemCameraInfo.screenHeight * spp) resultBuffer[threadIndex] = make_float4(0.0f);
	//if (sppLane == 0) resultBuffer[resultIndex] = make_float4(0.0f);
	if (sppLane == 0) resultBuffer[resultIndex] *= ((float)groupFrameCount - 1) / (float)groupFrameCount;
	__syncthreads();

	uint32_t randomNumberSeed = groupRandomNumberSeed + threadIndex;
	uint2 seed2 = pcg2d(make_uint2(threadIndex) * (sppLane * 10 + spp * randomNumberSeed + 1));
	randomNumberSeed = seed2.x + seed2.y;

	glm::vec3 radiance = glm::vec3(0.0f, 0.0f, 0.0f);
	float RR = 0.8f;
	float pdf = 1.0f;
	glm::vec3 bsdf = glm::vec3(1.0f);
	bool hit = true;
	FzbTriangleAttribute hitTriangleAttribute;
	FzbTriangleAttribute lastHitTriangleAttribute;

	glm::vec2 texelXY = glm::vec2(resultIndex % groupCameraInfo.screenWidth, resultIndex / groupCameraInfo.screenWidth);
	glm::vec4 screenPos = glm::vec4(((texelXY + Hammersley(sppLane, spp)) / glm::vec2(groupCameraInfo.screenWidth, groupCameraInfo.screenHeight)) * 2.0f - 1.0f, 0.0f, 1.0f);	//vulkan�н�ƽ��ndcDepth��[0,1]
	screenPos = groupCameraInfo.inversePVMatrix * screenPos;
	screenPos /= screenPos.w;
	FzbRay ray;
	ray.startPos = groupCameraInfo.cameraWorldPos;
	ray.direction = glm::normalize(glm::vec3(screenPos) - ray.startPos);
	ray.depth = FLT_MAX;
	ray.refraction = false;
	ray.ext = true;
	glm::vec3 lastDirection = -ray.direction;

	hit = sceneCollisionDetection(bvhNodeArray, bvhTriangleInfoArray, vertices, materialTextures, ray, hitTriangleAttribute);
	if (!hit) return;
	radiance += getRadiance(hitTriangleAttribute, ray, &lightSet, vertices, materialTextures, bvhNodeArray, bvhTriangleInfoArray, randomNumberSeed, groupSetting.useSphericalRectangleSample);

	uint32_t maxBonceDepth = 2;
#pragma nounroll
	while (maxBonceDepth > 0) {
		randomNumberSeed += maxBonceDepth;
		float randomNumber = rand(randomNumberSeed);
		if (randomNumber > RR) break;
		pdf *= RR;

		lastDirection = -ray.direction;
		//generateRay(hitTriangleAttribute, pdf, ray, randomNumberSeed);
		generateRay_SVOPathGuiding(hitTriangleAttribute, pdf, ray, randomNumberSeed, groupSetting);

		lastHitTriangleAttribute = hitTriangleAttribute;
		hit = sceneCollisionDetection(bvhNodeArray, bvhTriangleInfoArray, vertices, materialTextures, ray, hitTriangleAttribute);
		if (!hit) break;
		
		bsdf *= getBSDF(lastHitTriangleAttribute, ray.direction, lastDirection, ray) * glm::abs(glm::dot(ray.direction, lastHitTriangleAttribute.normal));
		radiance += getRadiance(hitTriangleAttribute, ray, &lightSet, vertices, materialTextures, bvhNodeArray, bvhTriangleInfoArray, randomNumberSeed, groupSetting.useSphericalRectangleSample) * bsdf / pdf;
		--maxBonceDepth;
	}

	radiance /= spp;
	radiance /= groupFrameCount;
	radiance = glm::min(radiance, glm::vec3(1.0f));
	if (useExternSharedMemory && threadIdx.x < groupSppIndex * spp) {
		//�����������sppΪ32�������������������warp�д���
		atomicAdd(&groupResultRadiance[groupSppIndex].x, radiance.x);
		atomicAdd(&groupResultRadiance[groupSppIndex].y, radiance.y);
		atomicAdd(&groupResultRadiance[groupSppIndex].z, radiance.z);
	}
	else {
		atomicAdd(&resultBuffer[resultIndex].x, radiance.x);
		atomicAdd(&resultBuffer[resultIndex].y, radiance.y);
		atomicAdd(&resultBuffer[resultIndex].z, radiance.z);
		return;
	}
	__syncthreads();
	if (useExternSharedMemory) {
		if (sppLane == 0) {
			atomicAdd(&resultBuffer[resultIndex].x, groupResultRadiance[groupSppIndex].x);
			atomicAdd(&resultBuffer[resultIndex].y, groupResultRadiance[groupSppIndex].y);
			atomicAdd(&resultBuffer[resultIndex].z, groupResultRadiance[groupSppIndex].z);
		}
	}
}
//-----------------------------------------------------------------------------------------------------------------
FzbSVOPathGuidingCuda::FzbSVOPathGuidingCuda() {};
FzbSVOPathGuidingCuda::FzbSVOPathGuidingCuda(std::shared_ptr<FzbRayTracingSourceManager_Cuda> sourceManager, FzbSVOPathGuidingCudaSetting setting, std::shared_ptr<FzbSVOCuda_PG> SVOCuda_PG) {
	if (getCudaDeviceForVulkanPhysicalDevice(FzbRenderer::globalData.physicalDevice) == hipInvalidDeviceId) {
		throw std::runtime_error("CUDA��Vulkan�õĲ���ͬһ��GPU������");
	}
	this->sourceManager = sourceManager;
	this->setting = setting;
	this->SVOCuda_PG = SVOCuda_PG;

	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(svoPathGuidingSetting), &setting, sizeof(FzbSVOPathGuidingCudaSetting)));

	//����cuda���ã������ʹ��L1 cache
	hipFuncSetAttribute(reinterpret_cast<const void*>(svoPathGuiding_cuda<true>), hipFuncAttributeMaxDynamicSharedMemorySize, (setting.spp >= sharedMemorySPP ? blockSize / setting.spp : 0) * sizeof(float3));	//3070 128KB����L1 96KB��sharedMemory 32KB
	hipFuncSetAttribute(reinterpret_cast<const void*>(svoPathGuiding_cuda<false>), hipFuncAttributeMaxDynamicSharedMemorySize, 0);
}
void FzbSVOPathGuidingCuda::SVOPathGuiding(HANDLE startSemaphoreHandle) {
	this->sourceManager->createRuntimeSource();

	VkExtent2D resolution = FzbRenderer::globalData.getResolution();
	uint32_t texelCount = resolution.width * resolution.height;
	uint32_t rayCount = texelCount * setting.spp;
	uint32_t gridSize = (rayCount + blockSize - 1) / blockSize;

	uint32_t sharedMemorySize = (setting.spp >= sharedMemorySPP ? blockSize / setting.spp : 0) * sizeof(float3);

	if (startSemaphoreHandle) {
		if (!this->extStartSemphores.count(startSemaphoreHandle))
			this->extStartSemphores.insert({ startSemaphoreHandle, importVulkanSemaphoreObjectFromNTHandle(startSemaphoreHandle) });
		CHECK(waitExternalSemaphore(this->extStartSemphores[startSemaphoreHandle], sourceManager->stream));
	}

	//CHECK(hipDeviceSynchronize());
	//double start = cpuSecond();

	if (setting.spp >= sharedMemorySPP) 
		svoPathGuiding_cuda<true> << <gridSize, blockSize, sharedMemorySize, sourceManager->stream >> >
		(
			sourceManager->resultBuffer, sourceManager->vertices, sourceManager->materialTextures,
			sourceManager->bvhNodeArray, sourceManager->bvhTriangleInfoArray, rayCount
		);
	else
		svoPathGuiding_cuda<false> << <gridSize, blockSize, sharedMemorySize, sourceManager->stream >> > 
		(
			sourceManager->resultBuffer, sourceManager->vertices, sourceManager->materialTextures,
			sourceManager->bvhNodeArray, sourceManager->bvhTriangleInfoArray, rayCount
		);
	//checkKernelFunction();
	//CHECK(hipDeviceSynchronize());
	//this->meanRunTime += cpuSecond() - start;
	//++runCount;
	//if (runCount == 20) {
	//	std::cout << meanRunTime / runCount << std::endl;
	//	runCount = 0;
	//	meanRunTime = 0.0;
	//}

	signalExternalSemaphore(sourceManager->extRayTracingFinishedSemaphore, sourceManager->stream);
}
void FzbSVOPathGuidingCuda::clean() {
	for (auto& pair : this->extStartSemphores) CHECK(hipDestroyExternalSemaphore(pair.second));
}
