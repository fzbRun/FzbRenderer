#include "./PathTracing_CUDA.cuh"

FzbPathTracingCuda::FzbPathTracingCuda() {};
FzbPathTracingCuda::FzbPathTracingCuda(VkPhysicalDevice vkPhysicalDevice, FzbScene* scene, FzbImage pathTracingResultMap, HANDLE pathTracingFinishedSemphoreHandle, std::vector<HANDLE> startSemaphoreHandles) {
	uint32_t resultMapSize = pathTracingResultMap.width * pathTracingResultMap.height * sizeof(uint32_t);		//SRGB��uint32_t��С��ͬ
	fromVulkanImageToCudaSurface(vkPhysicalDevice, pathTracingResultMap, pathTracingResultMap.handle, resultMapSize, false, resultMapExtMem, resultMapMipmap, resultMapObject);

	extPathTracingFinishedSemaphore = importVulkanSemaphoreObjectFromNTHandle(pathTracingFinishedSemphoreHandle);
	startSemaphoreNum = startSemaphoreHandles.size();
	extStartSemaphores.resize(startSemaphoreNum);
	for (int i = 0; i < startSemaphoreNum; ++i) extStartSemaphores[i] = importVulkanSemaphoreObjectFromNTHandle(startSemaphoreHandles[i]);

	vertexExtMem = importVulkanMemoryObjectFromNTHandle(scene->vertexBuffer.handle, scene->vertexBuffer.size, false);
	vertices = (float*)mapBufferOntoExternalMemory(vertexExtMem, 0, scene->vertexBuffer.size);

	for (auto& sceneImage : scene->sceneImages) {
		FzbImage& texture = sceneImage.second;
		uint32_t textureSize = texture.width * texture.height * sizeof(uint32_t);

	}
	
	CHECK(hipStreamCreate(&stream));
}
void FzbPathTracingCuda::pathTracing(){
	for (int i = 0; i < startSemaphoreNum; ++i) waitExternalSemaphore(extStartSemaphores[i], stream);
}

void FzbPathTracingCuda::clean() {
	CHECK(hipDestroySurfaceObject(resultMapObject));
	CHECK(hipFreeMipmappedArray(resultMapMipmap));
	CHECK(hipDestroyExternalMemory(resultMapExtMem));
	CHECK(hipDestroyExternalSemaphore(extPathTracingFinishedSemaphore));
	for (int i = 0; i < startSemaphoreNum; ++i) CHECK(hipDestroyExternalSemaphore(extStartSemaphores[i]));
	CHECK(hipDestroyExternalMemory(vertexExtMem));
	CHECK(hipFree(vertices));
	CHECK(hipStreamDestroy(stream));
}