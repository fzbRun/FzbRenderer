#pragma once

#include "vulkanCudaInterop.cuh"

#ifndef VULKAN_CUDA_INTEROP_CU
#define VULKAN_CUDA_INTEROP_CU

double cpuSecond() {
    auto now = std::chrono::high_resolution_clock::now();
    return std::chrono::duration<double>(now.time_since_epoch()).count();
}

//------------------------------------------------------------Vulkan������������-----------------------------------------------------------------

/*
When importing memory and synchronization objects exported by Vulkan, they must be imported and mapped on the same device as they were created on. 
The CUDA device that corresponds to the Vulkan physical device on which the objects were created can be determined by comparing the UUID of a CUDA device with that of the Vulkan physical device, 
as shown in the following code sample. Note that the Vulkan physical device should not be part of a device group that contains more than one Vulkan physical device. 
The device group as returned by vkEnumeratePhysicalDeviceGroups that contains the given Vulkan physical device must have a physical device count of 1.
*/
int getCudaDeviceForVulkanPhysicalDevice(VkPhysicalDevice vkPhysicalDevice) {

    VkPhysicalDeviceIDProperties vkPhysicalDeviceIDProperties = {};
    vkPhysicalDeviceIDProperties.sType = VK_STRUCTURE_TYPE_PHYSICAL_DEVICE_ID_PROPERTIES;
    vkPhysicalDeviceIDProperties.pNext = NULL;

    VkPhysicalDeviceProperties2 vkPhysicalDeviceProperties2 = {};
    vkPhysicalDeviceProperties2.sType = VK_STRUCTURE_TYPE_PHYSICAL_DEVICE_PROPERTIES_2;
    vkPhysicalDeviceProperties2.pNext = &vkPhysicalDeviceIDProperties;

    vkGetPhysicalDeviceProperties2(vkPhysicalDevice, &vkPhysicalDeviceProperties2);

    int cudaDeviceCount;
    hipGetDeviceCount(&cudaDeviceCount);

    for (int cudaDevice = 0; cudaDevice < cudaDeviceCount; cudaDevice++) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, cudaDevice);
        if (!memcmp(&deviceProp.uuid, vkPhysicalDeviceIDProperties.deviceUUID, VK_UUID_SIZE)) {
            return cudaDevice;
        }
    }

    return hipInvalidDeviceId;

}

/*
On Linux and Windows 10, both dedicated and non-dedicated memory objects exported by Vulkan can be imported into CUDA. 
On Windows 7, only dedicated memory objects can be imported. 
When importing a Vulkan dedicated memory object, the flag hipExternalMemoryDedicated must be set.
*/

/*
A Vulkan memory object exported using VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_FD_BIT can be imported into CUDA using the file descriptor associated with that object as shown below.
Note that CUDA assumes ownership of the file descriptor once it is imported. 
Using the file descriptor after a successful import results in undefined behavior.
*/
hipExternalMemory_t importVulkanMemoryObjectFromFileDescriptor(int fd, unsigned long long size, bool isDedicated) {

    hipExternalMemory_t extMem = NULL;
    hipExternalMemoryHandleDesc desc = {};

    memset(&desc, 0, sizeof(desc));

    desc.type = hipExternalMemoryHandleTypeOpaqueFd;
    desc.handle.fd = fd;
    desc.size = size;
    if (isDedicated) {
        desc.flags |= hipExternalMemoryDedicated;
    }

    // Input parameter 'fd' should not be used beyond this point as CUDA has assumed ownership of it
    hipImportExternalMemory(&extMem, &desc);

    return extMem;

}

/*
A Vulkan memory object exported using VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT can be imported into CUDA using the NT handle associated with that object as shown below.
Note that CUDA does not assume ownership of the NT handle and it is the application��s responsibility to close the handle when it is not required anymore. 
The NT handle holds a reference to the resource, so it must be explicitly freed before the underlying memory can be freed.
*/
hipExternalMemory_t importVulkanMemoryObjectFromNTHandle(HANDLE handle, unsigned long long size, bool isDedicated) {

    hipExternalMemory_t extMem = NULL;
    hipExternalMemoryHandleDesc desc = {};

    memset(&desc, 0, sizeof(desc));

    desc.type = hipExternalMemoryHandleTypeOpaqueWin32;
    desc.handle.win32.handle = handle;
    desc.size = size;
    if (isDedicated) {
        desc.flags |= hipExternalMemoryDedicated;
    }

    CHECK(hipImportExternalMemory(&extMem, &desc));

    // Input parameter 'handle' should be closed if it's not needed anymore
    //CloseHandle(handle);

    return extMem;

}

/*
A Vulkan memory object exported using VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_BIT can also be imported using a named handle if one exists as shown below.
*/
hipExternalMemory_t importVulkanMemoryObjectFromNamedNTHandle(LPCWSTR name, unsigned long long size, bool isDedicated) {
    hipExternalMemory_t extMem = NULL;
    hipExternalMemoryHandleDesc desc = {};

    memset(&desc, 0, sizeof(desc));

    desc.type = hipExternalMemoryHandleTypeOpaqueWin32;
    desc.handle.win32.name = (void*)name;
    desc.size = size;
    if (isDedicated) {
        desc.flags |= hipExternalMemoryDedicated;
    }

    hipImportExternalMemory(&extMem, &desc);

    return extMem;
}

/*
A Vulkan memory object exported using VK_EXTERNAL_MEMORY_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT can be imported into CUDA using the globally shared D3DKMT handle 
associated with that object as shown below. 
Since a globally shared D3DKMT handle does not hold a reference to the underlying memory it is automatically destroyed when all other references to the resource are destroyed.
*/
hipExternalMemory_t importVulkanMemoryObjectFromKMTHandle(HANDLE handle, unsigned long long size, bool isDedicated) {
    hipExternalMemory_t extMem = NULL;
    hipExternalMemoryHandleDesc desc = {};

    memset(&desc, 0, sizeof(desc));

    desc.type = hipExternalMemoryHandleTypeOpaqueWin32Kmt;
    desc.handle.win32.handle = (void*)handle;
    desc.size = size;
    if (isDedicated) {
        desc.flags |= hipExternalMemoryDedicated;
    }

    hipImportExternalMemory(&extMem, &desc);

    return extMem;
}

//----------------------------------------------------------------����������-----------------------------------------------------
/*
A device pointer can be mapped onto an imported memory object as shown below. 
The offset and size of the mapping must match that specified when creating the mapping using the corresponding Vulkan API.
All mapped device pointers must be freed using hipFree().
*/
/*
ò��ֻ�ܴ�vulkan�д������ݵ�cuda�������ܴ�cuda�д������ݵ�vulkan
����ֻ����vulkan�д���һ��buffer��Ȼ����cuda�н����copy��ȥ
*/
void* mapBufferOntoExternalMemory(hipExternalMemory_t extMem, unsigned long long offset, unsigned long long size) {

    void* ptr = NULL;

    hipExternalMemoryBufferDesc desc = {};
    memset(&desc, 0, sizeof(desc));
    desc.offset = offset;
    desc.size = size;

    hipExternalMemoryGetMappedBuffer(&ptr, extMem, &desc);

    // Note: ��ptr�� must eventually be freed using hipFree()
    return ptr;

}

//----------------------------------------------------------------����------------------------------------------------------------
/*
A CUDA mipmapped array can be mapped onto an imported memory object as shown below. 
The offset, dimensions, format and number of mip levels must match that specified when creating the mapping using the corresponding Vulkan API. 
Additionally, if the mipmapped array is bound as a color target in Vulkan, the flagcudaArrayColorAttachment must be set. 
All mapped mipmapped arrays must be freed using hipFreeMipmappedArray(). 
The following code sample shows how to convert Vulkan parameters into the corresponding CUDA parameters when mapping mipmapped arrays onto imported memory objects.
*/
//ò��û�в���mipmap�ģ��ҿ�nvida��runtime API�ĵ���
hipMipmappedArray_t mapMipmappedArrayOntoExternalMemory(hipExternalMemory_t extMem, unsigned long long offset, hipChannelFormatDesc* formatDesc, hipExtent* extent, unsigned int flags, unsigned int numLevels) {

    hipMipmappedArray_t mipmap = NULL;
    cudaExternalMemoryMipmappedArrayDesc desc = {};

    memset(&desc, 0, sizeof(desc));

    desc.offset = offset;
    desc.formatDesc = *formatDesc;
    desc.extent = *extent;
    desc.flags = flags;
    desc.numLevels = numLevels;

    // Note: 'mipmap' must eventually be freed using hipFreeMipmappedArray()
    CHECK(cudaExternalMemoryGetMappedMipmappedArray(&mipmap, extMem, &desc));

    return mipmap;

}

hipChannelFormatDesc getCudaChannelFormatDescForVulkanFormat(VkFormat format)
{
    hipChannelFormatDesc d;

    memset(&d, 0, sizeof(d));

    switch (format) {
        case VK_FORMAT_R8_UINT:             d.x = 8;  d.y = 0;  d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R8_SINT:             d.x = 8;  d.y = 0;  d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindSigned;   break;
        case VK_FORMAT_R8G8_UINT:           d.x = 8;  d.y = 8;  d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R8G8_SINT:           d.x = 8;  d.y = 8;  d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindSigned;   break;
        case VK_FORMAT_R8G8B8A8_UINT:       d.x = 8;  d.y = 8;  d.z = 8;  d.w = 8;  d.f = hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R8G8B8A8_SINT:       d.x = 8;  d.y = 8;  d.z = 8;  d.w = 8;  d.f = hipChannelFormatKindSigned;   break;
        case VK_FORMAT_R16_UINT:            d.x = 16; d.y = 0;  d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R16_SINT:            d.x = 16; d.y = 0;  d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindSigned;   break;
        case VK_FORMAT_R16G16_UINT:         d.x = 16; d.y = 16; d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R16G16_SINT:         d.x = 16; d.y = 16; d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindSigned;   break;
        case VK_FORMAT_R16G16B16A16_UINT:   d.x = 16; d.y = 16; d.z = 16; d.w = 16; d.f = hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R16G16B16A16_SINT:   d.x = 16; d.y = 16; d.z = 16; d.w = 16; d.f = hipChannelFormatKindSigned;   break;
        case VK_FORMAT_R32_UINT:            d.x = 32; d.y = 0;  d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R32_SINT:            d.x = 32; d.y = 0;  d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindSigned;   break;
        case VK_FORMAT_R32_SFLOAT:          d.x = 32; d.y = 0;  d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindFloat;    break;
        case VK_FORMAT_R32G32_UINT:         d.x = 32; d.y = 32; d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R32G32_SINT:         d.x = 32; d.y = 32; d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindSigned;   break;
        case VK_FORMAT_R32G32_SFLOAT:       d.x = 32; d.y = 32; d.z = 0;  d.w = 0;  d.f = hipChannelFormatKindFloat;    break;
        case VK_FORMAT_R32G32B32A32_UINT:   d.x = 32; d.y = 32; d.z = 32; d.w = 32; d.f = hipChannelFormatKindUnsigned; break;
        case VK_FORMAT_R32G32B32A32_SINT:   d.x = 32; d.y = 32; d.z = 32; d.w = 32; d.f = hipChannelFormatKindSigned;   break;
        case VK_FORMAT_R32G32B32A32_SFLOAT: d.x = 32; d.y = 32; d.z = 32; d.w = 32; d.f = hipChannelFormatKindFloat;    break;
    default: assert(0);
    }

    return d;
}

hipExtent getCudaExtentForVulkanExtent(VkExtent3D vkExt, uint32_t arrayLayers, VkImageViewType vkImageViewType) {

    hipExtent e = { 0, 0, 0 };

    switch (vkImageViewType) {
    case VK_IMAGE_VIEW_TYPE_1D:         e.width = vkExt.width; e.height = 0;            e.depth = 0;           break;
    case VK_IMAGE_VIEW_TYPE_2D:         e.width = vkExt.width; e.height = vkExt.height; e.depth = 0;           break;
    case VK_IMAGE_VIEW_TYPE_3D:         e.width = vkExt.width; e.height = vkExt.height; e.depth = vkExt.depth; break;
    case VK_IMAGE_VIEW_TYPE_CUBE:       e.width = vkExt.width; e.height = vkExt.height; e.depth = arrayLayers; break;
    case VK_IMAGE_VIEW_TYPE_1D_ARRAY:   e.width = vkExt.width; e.height = 0;            e.depth = arrayLayers; break;
    case VK_IMAGE_VIEW_TYPE_2D_ARRAY:   e.width = vkExt.width; e.height = vkExt.height; e.depth = arrayLayers; break;
    case VK_IMAGE_VIEW_TYPE_CUBE_ARRAY: e.width = vkExt.width; e.height = vkExt.height; e.depth = arrayLayers; break;
    default: assert(0);
    }

    return e;
}

unsigned int getCudaMipmappedArrayFlagsForVulkanImage(VkImageViewType vkImageViewType, VkImageUsageFlags vkImageUsageFlags, bool allowSurfaceLoadStore) {

    unsigned int flags = 0;

    switch (vkImageViewType) {
        case VK_IMAGE_VIEW_TYPE_CUBE:       flags |= hipArrayCubemap;                    break;
        case VK_IMAGE_VIEW_TYPE_CUBE_ARRAY: flags |= hipArrayCubemap | hipArrayLayered; break;
        case VK_IMAGE_VIEW_TYPE_1D_ARRAY:   flags |= hipArrayLayered;                    break;
        case VK_IMAGE_VIEW_TYPE_2D_ARRAY:   flags |= hipArrayLayered;                    break;
        default: break;
    }

    if (vkImageUsageFlags & VK_IMAGE_USAGE_COLOR_ATTACHMENT_BIT) {
        flags |= cudaArrayColorAttachment;
    }

    if (allowSurfaceLoadStore) {
        flags |= hipArraySurfaceLoadStore;
    }

    return flags;

}

void fromVulkanImageToCudaTexture(VkPhysicalDevice vkPhysicalDevice, FzbImage& vkImage, HANDLE handle, unsigned long long size,
    bool isDedicated, hipExternalMemory_t& extMem, hipMipmappedArray_t& mipmap, hipTextureObject_t& texObj) {

    //���ж��Ƿ���ͬһ�������豸
    if (getCudaDeviceForVulkanPhysicalDevice(vkPhysicalDevice) == hipInvalidDeviceId) {
        throw std::runtime_error("CUDA��Vulkan�õĲ���ͬһ��GPU������");
    }

    //���Vulkan�������ڴ����
    extMem = importVulkanMemoryObjectFromNTHandle(handle, size, isDedicated);

    //������ӳ�䵽�ⲿ�ڴ����
    hipChannelFormatDesc format = getCudaChannelFormatDescForVulkanFormat(vkImage.format);
    hipExtent extent = getCudaExtentForVulkanExtent({ vkImage.width, vkImage.height, vkImage.depth }, vkImage.layerNum, vkImage.viewType);
    unsigned int flags = getCudaMipmappedArrayFlagsForVulkanImage(vkImage.viewType, vkImage.usage, false);   //hipArraySurfaceLoadStore��ʾ�Ƿ��д
    mipmap = mapMipmappedArrayOntoExternalMemory(extMem, 0, &format, &extent, flags, vkImage.mipLevels);    //hipMipmappedArray_t��ֻ����

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeMipmappedArray;
    resDesc.res.mipmap.mipmap = mipmap;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.addressMode[2] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModePoint;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;

    CHECK(hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL));

}

void fromVulkanImageToCudaSurface(VkPhysicalDevice vkPhysicalDevice, FzbImage& vkImage, HANDLE handle, unsigned long long size,
    bool isDedicated, hipExternalMemory_t& extMem, hipMipmappedArray_t& mipmap, hipSurfaceObject_t& surfObj) {

    //���ж��Ƿ���ͬһ�������豸
    if (getCudaDeviceForVulkanPhysicalDevice(vkPhysicalDevice) == hipInvalidDeviceId) {
        throw std::runtime_error("CUDA��Vulkan�õĲ���ͬһ��GPU������");
    }

    //���Vulkan�������ڴ����
    extMem = importVulkanMemoryObjectFromNTHandle(handle, size, isDedicated);

    //������ӳ�䵽�ⲿ�ڴ����
    hipChannelFormatDesc format = getCudaChannelFormatDescForVulkanFormat(vkImage.format);
    hipExtent extent = getCudaExtentForVulkanExtent({ vkImage.width, vkImage.height, vkImage.depth }, vkImage.layerNum, vkImage.viewType);
    unsigned int flags = getCudaMipmappedArrayFlagsForVulkanImage(vkImage.viewType, vkImage.usage, true);   //hipArraySurfaceLoadStore��ʾ�Ƿ��д
    mipmap = mapMipmappedArrayOntoExternalMemory(extMem, 0, &format, &extent, flags, vkImage.mipLevels);

    hipArray_t cuArray;    //hipArray_t����ֱ���ں˺����ж�д������ԭ������
    CHECK(hipGetMipmappedArrayLevel(&cuArray, mipmap, 0)); // ѡ�� Mipmap �㼶

    hipResourceDesc resDesc = {};
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;

    CHECK(hipCreateSurfaceObject(&surfObj, &resDesc));

}

//-------------------------------------------------------------�ź���---------------------------------------------------------
/*
A Vulkan semaphore object exported using VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_FD_BITcan be imported into CUDA using the file descriptor associated with that object as shown below.
Note that CUDA assumes ownership of the file descriptor once it is imported.
Using the file descriptor after a successful import results in undefined behavior.
*/
hipExternalSemaphore_t importVulkanSemaphoreObjectFromFileDescriptor(int fd) {

    hipExternalSemaphore_t extSem = NULL;
    hipExternalSemaphoreHandleDesc desc = {};

    memset(&desc, 0, sizeof(desc));

    desc.type = hipExternalSemaphoreHandleTypeOpaqueFd;
    desc.handle.fd = fd;

    hipImportExternalSemaphore(&extSem, &desc);

    // Input parameter 'fd' should not be used beyond this point as CUDA has assumed ownership of it
    return extSem;

}

/*
A Vulkan semaphore object exported using VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT can be imported into CUDA using the NT handle associated with that object as shown below.
Note that CUDA does not assume ownership of the NT handle and it is the application��s responsibility to close the handle when it is not required anymore.
The NT handle holds a reference to the resource, so it must be explicitly freed before the underlying semaphore can be freed.
*/
hipExternalSemaphore_t importVulkanSemaphoreObjectFromNTHandle(HANDLE handle) {

    hipExternalSemaphore_t extSem = NULL;
    hipExternalSemaphoreHandleDesc desc = {};

    memset(&desc, 0, sizeof(desc));

    desc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;
    desc.handle.win32.handle = handle;

    hipImportExternalSemaphore(&extSem, &desc);

    // Input parameter 'handle' should be closed if it's not needed anymore
    //CloseHandle(handle);

    return extSem;
}

/*
A Vulkan semaphore object exported using VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_BIT can also be imported using a named handle if one exists as shown below.
*/
hipExternalSemaphore_t importVulkanSemaphoreObjectFromNamedNTHandle(LPCWSTR name) {

    hipExternalSemaphore_t extSem = NULL;
    hipExternalSemaphoreHandleDesc desc = {};

    memset(&desc, 0, sizeof(desc));

    desc.type = hipExternalSemaphoreHandleTypeOpaqueWin32;
    desc.handle.win32.name = (void*)name;

    hipImportExternalSemaphore(&extSem, &desc);

    return extSem;
}

/*
A Vulkan semaphore object exported using VK_EXTERNAL_SEMAPHORE_HANDLE_TYPE_OPAQUE_WIN32_KMT_BIT can be imported into CUDA using the globally shared D3DKMT handle
associated with that object as shown below.
Since a globally shared D3DKMT handle does not hold a reference to the underlying semaphore it is automatically destroyed when all other references to the resource are destroyed.
*/
hipExternalSemaphore_t importVulkanSemaphoreObjectFromKMTHandle(HANDLE handle) {

    hipExternalSemaphore_t extSem = NULL;
    hipExternalSemaphoreHandleDesc desc = {};

    memset(&desc, 0, sizeof(desc));

    desc.type = hipExternalSemaphoreHandleTypeOpaqueWin32Kmt;
    desc.handle.win32.handle = (void*)handle;

    hipImportExternalSemaphore(&extSem, &desc);

    return extSem;

}

/*
An imported Vulkan semaphore object can be signaled as shown below.
Signaling such a semaphore object sets it to the signaled state.
The corresponding wait that waits on this signal must be issued in Vulkan.
Additionally, the wait that waits on this signal must be issued after this signal has been issued.
*/
void signalExternalSemaphore(hipExternalSemaphore_t extSem, hipStream_t stream) {

    hipExternalSemaphoreSignalParams params = {};

    memset(&params, 0, sizeof(params));

    hipSignalExternalSemaphoresAsync(&extSem, &params, 1, stream);

}

/*
An imported Vulkan semaphore object can be waited on as shown below.
Waiting on such a semaphore object waits until it reaches the signaled state and then resets it back to the unsignaled state.
The corresponding signal that this wait is waiting on must be issued in Vulkan.
Additionally, the signal must be issued before this wait can be issued.
*/
void waitExternalSemaphore(hipExternalSemaphore_t extSem, hipStream_t stream) {

    hipExternalSemaphoreWaitParams params = {};

    memset(&params, 0, sizeof(params));

    hipWaitExternalSemaphoresAsync(&extSem, &params, 1, stream);

}

#endif
