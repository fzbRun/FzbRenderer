#include "hip/hip_runtime.h"
#include "FzbSVOCuda_PG.cuh"
#include "../../../RayTracing/CUDA/FzbCollisionDetection.cuh"

__global__ void getSVONodesWeight_device(
	float* SVONodeWeights,
	const FzbSVOIndivisibleNodeInfo* __restrict__ indivisibleNodeInfos,
	uint32_t SVONodeCountInLayer, uint32_t SVONodeTotalCount, uint32_t SVOInDivisibleNodeTotalCount,
	FzbSVONodeData_PG** SVONodes,
	float* SVODivisibleNodeBlockWeight, uint32_t divisibleNodeOffset, uint32_t fatherDivisibleNodeOffset,
	uint32_t maxSVOLayer, uint32_t targetNodeLayer, uint32_t layerNodeOffset,
	const float* __restrict__ vertices, const FzbBvhNode* __restrict__ bvhNodeArray, const FzbBvhNodeTriangleInfo* __restrict__ bvhTriangleInfoArray,
	const uint32_t testCount, const uint32_t threadBlockCountForOneNode
) {
	__shared__ FzbSVONodeData_PG* groupSVONodesArray[8];
	__shared__ uint32_t groupRandomNumberSeed;

	__shared__ FzbSVOIndivisibleNodeInfo nodeInfo;
	__shared__ FzbSVONodeData_PG nodeData;

	uint32_t indivisibleNodeIndex = blockIdx.x / threadBlockCountForOneNode;
	int targetNodeIndex = (blockIdx.x % threadBlockCountForOneNode) * blockDim.x + threadIdx.x;
	if (targetNodeIndex >= SVONodeCountInLayer) return;	//���ﲻ�᷵����SVONodes�е��̣߳����Բ���Ժ���ϴ�����Ӱ��

	if (threadIdx.x < maxSVOLayer) groupSVONodesArray[threadIdx.x] = SVONodes[threadIdx.x]; //�߳���������Ϊ8����Ϊ��һ�����8��
	if (threadIdx.x == 0) {
		groupRandomNumberSeed = systemRandomNumberSeed;

		nodeInfo = indivisibleNodeInfos[indivisibleNodeIndex];
		nodeData = groupSVONodesArray[nodeInfo.nodeLayer][nodeInfo.nodeIndex];
	}
	__syncthreads();

	indivisibleNodeIndex = nodeData.label - 1;	//����SVO�еڼ������ɷ�node

	uint32_t hitCount = 0;
	FzbSVONodeData_PG targetNodeData = groupSVONodesArray[targetNodeLayer][targetNodeIndex];
	bool hasData = targetNodeData.irradiance.x != 0 || targetNodeData.irradiance.y != 0 || targetNodeData.irradiance.z != 0;
	float weight = 0.0f;
	if (hasData) {
		if (targetNodeData.indivisible == 0) {
			uint32_t weightIndex = indivisibleNodeIndex * SVONodeTotalCount + divisibleNodeOffset + targetNodeData.label - 1;
			weight = SVODivisibleNodeBlockWeight[weightIndex];
		}
		else if (!(targetNodeLayer == nodeInfo.nodeLayer && targetNodeIndex == nodeInfo.nodeIndex)) {
			bool isFather =
				targetNodeData.AABB.leftX <= nodeData.AABB.leftX &&
				targetNodeData.AABB.leftY <= nodeData.AABB.leftY &&
				targetNodeData.AABB.leftZ <= nodeData.AABB.leftZ &&
				targetNodeData.AABB.rightX >= nodeData.AABB.rightX &&
				targetNodeData.AABB.rightY >= nodeData.AABB.rightY &&
				targetNodeData.AABB.rightZ >= nodeData.AABB.rightZ;

			uint32_t randomNumberSeed = groupRandomNumberSeed + threadIdx.x + blockDim.x * blockIdx.x;
			float distanceX = nodeData.AABB.rightX - nodeData.AABB.leftX;
			float distanceY = nodeData.AABB.rightY - nodeData.AABB.leftY;
			float distanceZ = nodeData.AABB.rightZ - nodeData.AABB.leftZ;

			float targetDistanceX = targetNodeData.AABB.rightX - targetNodeData.AABB.leftX;
			float targetDistanceY = targetNodeData.AABB.rightY - targetNodeData.AABB.leftY;
			float targetDistanceZ = targetNodeData.AABB.rightZ - targetNodeData.AABB.leftZ;
			FzbRay ray;
			FzbTriangleAttribute triangleAttribute;
			if (isFather) {
				for (int i = 0; i < testCount; ++i) {
					ray.depth = FLT_MAX;

					uint32_t faceIndex = uint32_t(rand(randomNumberSeed) * 6);	//0��ʾleftX��1��ʾrightX����
					float randomU = rand(randomNumberSeed);		//��ǰnode��AABB�ϵ������
					float randomV = rand(randomNumberSeed);
					ray.startPos = glm::vec3(nodeData.AABB.leftX, nodeData.AABB.leftY, nodeData.AABB.leftZ);
					if (faceIndex & 4) {
						ray.startPos.z += (faceIndex & 1) * distanceZ;
						ray.startPos.x += randomU * distanceX;
						ray.startPos.y += randomV * distanceY;
					}
					else if (faceIndex & 2) {
						ray.startPos.y += (faceIndex & 1) * distanceY;
						ray.startPos.x += randomU * distanceX;
						ray.startPos.z += randomV * distanceZ;
					}
					else {
						ray.startPos.x += (faceIndex & 1) * distanceX;
						ray.startPos.z += randomU * distanceZ;
						ray.startPos.y += randomV * distanceY;
					}

					faceIndex = uint32_t(rand(randomNumberSeed) * 6);
					randomU = rand(randomNumberSeed);		//targetnode��AABB�ϵ������
					randomV = rand(randomNumberSeed);
					ray.direction = glm::vec3(targetNodeData.AABB.leftX, targetNodeData.AABB.leftY, targetNodeData.AABB.leftZ);
					if (faceIndex & 4) {
						ray.direction.z += (faceIndex & 1) * targetDistanceZ;
						ray.direction.x += randomU * targetDistanceX;
						ray.direction.y += randomV * targetDistanceY;
					}
					else if (faceIndex & 2) {
						ray.direction.y += (faceIndex & 1) * targetDistanceY;
						ray.direction.x += randomU * targetDistanceX;
						ray.direction.z += randomV * targetDistanceZ;
					}
					else {
						ray.direction.x += (faceIndex & 1) * targetDistanceX;
						ray.direction.z += randomU * targetDistanceZ;
						ray.direction.y += randomV * targetDistanceY;
					}

					ray.direction = ray.direction - ray.startPos;
					float r = glm::length(ray.direction);
					ray.direction = glm::normalize(ray.direction);

					bool hit = sceneCollisionDetection(bvhNodeArray, bvhTriangleInfoArray, vertices, nullptr, ray, triangleAttribute, false);
					if (!hit) continue;
					if (ray.hitPos.x < targetNodeData.AABB.leftX || ray.hitPos.y < targetNodeData.AABB.leftY || ray.hitPos.z < targetNodeData.AABB.leftZ ||
						ray.hitPos.x > targetNodeData.AABB.rightX || ray.hitPos.y > targetNodeData.AABB.rightY || ray.hitPos.z > targetNodeData.AABB.rightZ) continue;
					++hitCount;
				}
			}
			else {
				glm::vec3 nodeCenterPos = glm::vec3(nodeData.AABB.leftX + nodeData.AABB.rightX, nodeData.AABB.leftY + nodeData.AABB.rightY, nodeData.AABB.leftZ + nodeData.AABB.rightZ) * 0.5f;
				glm::vec3 targetNodeCenterPos = glm::vec3(targetNodeData.AABB.leftX + targetNodeData.AABB.rightX, targetNodeData.AABB.leftY + targetNodeData.AABB.rightY, targetNodeData.AABB.leftZ + targetNodeData.AABB.rightZ) * 0.5f;
				glm::vec3 nodeDirection = targetNodeCenterPos - nodeCenterPos;

				for (int i = 0; i < testCount; ++i) {
					ray.depth = FLT_MAX;

					uint32_t faceIndex = uint32_t(rand(randomNumberSeed) * 6);	//0��ʾleftX��1��ʾrightX����
					float randomU = rand(randomNumberSeed);		//��ǰnode��AABB�ϵ������
					float randomV = rand(randomNumberSeed);
					ray.startPos = glm::vec3(nodeData.AABB.leftX, nodeData.AABB.leftY, nodeData.AABB.leftZ);
					if (faceIndex & 4) {
						ray.startPos.z += (faceIndex & 1) * distanceZ;
						ray.startPos.x += randomU * distanceX;
						ray.startPos.y += randomV * distanceY;
					}
					else if (faceIndex & 2) {
						ray.startPos.y += (faceIndex & 1) * distanceY;
						ray.startPos.x += randomU * distanceX;
						ray.startPos.z += randomV * distanceZ;
					}
					else {
						ray.startPos.x += (faceIndex & 1) * distanceX;
						ray.startPos.z += randomU * distanceZ;
						ray.startPos.y += randomV * distanceY;
					}

					faceIndex = uint32_t(rand(randomNumberSeed) * 3);
					randomU = rand(randomNumberSeed);		//targetnode��AABB�ϵ������
					randomV = rand(randomNumberSeed);
					ray.direction = glm::vec3(targetNodeData.AABB.leftX, targetNodeData.AABB.leftY, targetNodeData.AABB.leftZ);

					if (faceIndex & 2) {
						ray.direction.z += nodeDirection.z < 0 ? targetDistanceZ : 0.0f;	//�ں��
						ray.direction.x += randomU * targetDistanceX;
						ray.direction.y += randomV * targetDistanceY;
					}
					else if (faceIndex & 1) {
						ray.direction.y += nodeDirection.y < 0 ? targetDistanceY : 0.0f;	//���±�
						ray.direction.x += randomU * targetDistanceX;
						ray.direction.z += randomV * targetDistanceZ;
					}
					else {
						ray.direction.x += nodeDirection.x < 0 ? targetDistanceX : 0.0f;	//�����
						ray.direction.z += randomU * targetDistanceZ;
						ray.direction.y += randomV * targetDistanceY;
					}

					ray.direction = ray.direction - ray.startPos;
					float r = glm::length(ray.direction);
					ray.direction = glm::normalize(ray.direction);

					bool hit = sceneCollisionDetection(bvhNodeArray, bvhTriangleInfoArray, vertices, nullptr, ray, triangleAttribute, false);
					if (!hit) continue;
					if (ray.hitPos.x < targetNodeData.AABB.leftX || ray.hitPos.y < targetNodeData.AABB.leftY || ray.hitPos.z < targetNodeData.AABB.leftZ ||
						ray.hitPos.x > targetNodeData.AABB.rightX || ray.hitPos.y > targetNodeData.AABB.rightY || ray.hitPos.z > targetNodeData.AABB.rightZ) continue;
					++hitCount;
				}
			}
			float occlusionRatio = (float)hitCount / testCount;
			weight = occlusionRatio * glm::length(targetNodeData.irradiance);
		}
	}

	uint32_t warpLane = threadIdx.x & 31;
	uint32_t firstBlockLane = (warpLane / 8) << 3;
	float blockWeightSum = weight;
	for (int offset = 4; offset > 0; offset /= 2)
		blockWeightSum += __shfl_down_sync(0xFFFFFFFF, blockWeightSum, offset);
	blockWeightSum = __shfl_sync(0xFFFFFFFF, blockWeightSum, firstBlockLane);

	if (weight > 0.0f) {
		uint32_t weightIndex = indivisibleNodeIndex * SVONodeTotalCount + layerNodeOffset + targetNodeIndex;
		SVONodeWeights[weightIndex] = weight / blockWeightSum;
	}
	if (warpLane == firstBlockLane && targetNodeLayer > 1) {
		uint32_t fatherNodeLabel = indivisibleNodeIndex * SVONodeTotalCount + fatherDivisibleNodeOffset + targetNodeIndex / 8;
		SVODivisibleNodeBlockWeight[fatherNodeLabel] = blockWeightSum;
	}
}
/*
__global__ void getSVONodesWeight_device_step2(
	float* SVONodeWeights, float* layerWeights,
	uint32_t SVONodeTotalCount, uint32_t SVOInDivisibleNodeTotalCount,
	FzbSVOLayerInfo* layerInfos, uint32_t maxSVOLayer
) {
	__shared__ FzbSVOLayerInfo groupSVOLayerInfos[8];
	uint32_t threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadIndex >= SVONodeTotalCount * SVOInDivisibleNodeTotalCount) return;

	if (blockIdx.x == gridDim.x - 1) {
		if(threadIdx.x == 0) for (int i = 0; i < maxSVOLayer; ++i) groupSVOLayerInfos[i] = layerInfos[i];
	}else if (threadIdx.x < maxSVOLayer) groupSVOLayerInfos[threadIdx.x] = layerInfos[threadIdx.x];
	uint32_t indivisibleNodeIndex = threadIndex / SVONodeTotalCount;
	int targetNodeIndex = threadIndex % SVONodeTotalCount;
	__syncthreads();

	int targetNodeLayer = 1;
	if (targetNodeIndex >= 8) {
		targetNodeIndex -= 8;
		while (targetNodeIndex >= 0) {
			++targetNodeLayer;
			targetNodeIndex -= groupSVOLayerInfos[targetNodeLayer - 1].divisibleNodeCount * 8;
		}
	}

	float weight = SVONodeWeights[threadIndex];
	uint32_t layerWeightIndex = indivisibleNodeIndex * maxSVOLayer + targetNodeLayer;
	float layerWeight = layerWeights[layerWeightIndex];
	SVONodeWeights[threadIndex] = layerWeight == 0.0f ? 0.0f : weight / layerWeight;
	//if (weight != 0.0f) printf("%f %f %f\n", weight, layerWeight, layerWeight == 0.0f ? 0.0f : weight / layerWeight);
}
*/

void FzbSVOCuda_PG::getSVONodesWeight() {
	uint32_t SVODivisibleNodeAccCount = 0;
	for (int i = 1; i < this->SVONodes_maxDepth - 1; ++i)	//Ҷ��divisibleNodeCount = 0
		SVODivisibleNodeAccCount += this->SVOLayerInfos_host[i].divisibleNodeCount;
	uint32_t layerDivisibleNodeOffset = SVODivisibleNodeAccCount;
	uint32_t layerNodeOffset = SVONodeTotalCount_host;
	for (int i = this->SVONodes_maxDepth - 1; i > 0; --i) {
		uint32_t layerNodeCount = SVOLayerInfos_host[i - 1].divisibleNodeCount * 8;		//��һ���node����
		uint32_t blockSize = layerNodeCount;	//�߳����СΪ�ò�node����
		uint32_t threadBlockCountForOneNode = (blockSize + 511) / 512;
		blockSize /= threadBlockCountForOneNode;
		while (((blockSize / threadBlockCountForOneNode) & 7) != 0) ++blockSize;
		uint32_t gridSize = SVOInDivisibleNodeTotalCount_host * threadBlockCountForOneNode;	//threadBlockCountForOneNode���߳������һ�����ɷ�node

		layerDivisibleNodeOffset -= this->SVOLayerInfos_host[i].divisibleNodeCount;		//�ò�Ŀɷ�node��ʼ����
		uint32_t fatherLayerDivisibleNodeOffset = layerDivisibleNodeOffset - this->SVOLayerInfos_host[i - 1].divisibleNodeCount;	//����ɷ�node��ʼ����
		layerNodeOffset -= layerNodeCount;		//�ò�node����ʼ����
		getSVONodesWeight_device<<<gridSize, blockSize, 0, stream>>>
		(
			SVONodeWeights, SVOIndivisibleNodeInfos,
			layerNodeCount, SVONodeTotalCount_host, SVOInDivisibleNodeTotalCount_host,
			SVONodes_multiLayer_Array,
			SVODivisibleNodeBlockWeight, layerDivisibleNodeOffset, fatherLayerDivisibleNodeOffset,
			SVONodes_maxDepth, i, layerNodeOffset,
			sourceManager->vertices, sourceManager->bvhNodeArray, sourceManager->bvhTriangleInfoArray,
			16, threadBlockCountForOneNode
		);
	}
	//uint32_t blockSize = SVONodeTotalCount_host;
	//uint32_t threadBlockCountForOneNode = (blockSize + 511) / 512;
	//blockSize /= threadBlockCountForOneNode;
	//while(((blockSize / threadBlockCountForOneNode) & 7) != 0) ++blockSize;
	//uint32_t gridSize = SVOInDivisibleNodeTotalCount_host * threadBlockCountForOneNode;
	//getSVONodesWeight_device <<<gridSize, blockSize, 0, stream>>>
	//(
	//	SVONodeWeights,
	//	SVOIndivisibleNodeInfos,
	//	SVONodeTotalCount_host, SVOInDivisibleNodeTotalCount_host,
	//	SVONodes_multiLayer_Array,
	//	SVOLayerInfos, SVONodes_maxDepth,
	//	sourceManager->vertices, sourceManager->bvhNodeArray, sourceManager->bvhTriangleInfoArray,
	//	16, threadBlockCountForOneNode
	//);
	//std::vector<float> layerWeights_host(1);
	//CHECK(hipMemcpy(layerWeights_host.data(), layerWeights + 57, layerWeights_host.size() * sizeof(float), hipMemcpyDeviceToHost));
	//std::cout << layerWeights_host[0] << std::endl;
	//getSVONodesWeight_device_step2<<<gridSize, blockSize, 0, stream>>>
	//(
	//	SVONodeWeights, layerWeights,
	//	SVONodeTotalCount_host, SVOInDivisibleNodeTotalCount_host,
	//	SVOLayerInfos, SVONodes_maxDepth
	//);
}

/*
Ϊÿ�����ɷ�node����weight����
weight�����СΪ��ֵnode�������������ɷ�node
*/
void FzbSVOCuda_PG::initGetSVONodesWeightSource() {
	//ÿ��node�����ָ��
	CHECK(hipMalloc((void**)&this->SVONodes_multiLayer_Array, this->SVONodes_maxDepth * sizeof(FzbSVONodeData_PG*)));
	CHECK(hipMemcpy(this->SVONodes_multiLayer_Array, SVONodes_multiLayer.data(), this->SVONodes_maxDepth * sizeof(FzbSVONodeData_PG*), hipMemcpyHostToDevice));

	CHECK(hipMalloc((void**)&this->SVODivisibleNodeBlockWeight, SVOIndivisibleNodeMaxCount * SVONodeMaxCount * sizeof(float)));
	CHECK(hipMemset(SVODivisibleNodeBlockWeight, 0, SVOIndivisibleNodeMaxCount * SVONodeMaxCount * sizeof(float)));

	//���յ�weiht��ÿ�����ɷ�node��Ӧÿ��SVONode(������ֵ)֮���weight
	CHECK(hipMalloc((void**)&this->SVONodeWeights, SVOIndivisibleNodeMaxCount * SVONodeMaxCount * sizeof(float)));
	CHECK(hipMemset(this->SVONodeWeights, 0, SVOIndivisibleNodeMaxCount * SVONodeMaxCount * sizeof(float)));
}