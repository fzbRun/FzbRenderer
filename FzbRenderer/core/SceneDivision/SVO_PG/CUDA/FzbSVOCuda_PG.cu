#include "hip/hip_runtime.h"
#include "FzbSVOCuda_PG.cuh"
#include "../../../common/FzbRenderer.h"
#include "../../../RayTracing/CUDA/FzbCollisionDetection.cuh"

//----------------------------------------------uniformBuffer--------------------------------------
__constant__ FzbVGBUniformData systemVGBUniformData;
__constant__ FzbSVOUnformData systemSVOUniformData;

const uint32_t createSVOKernelBlockSize = 512;
//----------------------------------------------�˺���--------------------------------------
__global__ void lightInject_cuda(FzbVoxelData_PG* VGB, const float* __restrict__ vertices, const hipTextureObject_t* __restrict__ materialTextures,
	const FzbBvhNode* __restrict__ bvhNodeArray, const FzbBvhNodeTriangleInfo* __restrict__ bvhTriangleInfoArray, const uint32_t rayCount);
//-------------------------------------------------------------------------------------------------
FzbSVOCuda_PG::FzbSVOCuda_PG() {};

__global__ void initSVO(FzbSVONodeData_PG* SVO, uint32_t svoCount) {
	uint32_t threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
	if (threadIndex >= svoCount) return;

	FzbSVONodeData_PG data;
	data.indivisible = 1;
	data.pdf = 1.0f;
	data.shuffleKey = 0;
	data.label = 0;
	data.AABB.leftX = FLT_MAX;
	data.AABB.leftY = FLT_MAX;
	data.AABB.leftZ = FLT_MAX;
	data.AABB.rightX = -FLT_MAX;
	data.AABB.rightY = -FLT_MAX;
	data.AABB.rightZ = -FLT_MAX;
	data.irradiance = glm::vec3(0.0f);
	SVO[threadIndex] = data;
}
FzbSVOCuda_PG::FzbSVOCuda_PG(std::shared_ptr<FzbRayTracingSourceManager_Cuda> sourceManager, FzbSVOSetting_PG setting, 
	FzbVGBUniformData VGBUniformData, FzbBuffer VGB, HANDLE SVOFinishedSemaphore_PG, FzbSVOUnformData SVOUniformData) {
	this->sourceManager = sourceManager;
	this->setting = setting;

	this->VGBUniformData = VGBUniformData;
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(systemVGBUniformData), &VGBUniformData, sizeof(FzbVGBUniformData)));

	this->SVOUniformData = SVOUniformData;
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(systemSVOUniformData), &SVOUniformData, sizeof(FzbSVOUnformData)));

	this->VGBExtMem = importVulkanMemoryObjectFromNTHandle(VGB.handle, VGB.size, false);
	this->VGB = (FzbVoxelData_PG*)mapBufferOntoExternalMemory(VGBExtMem, 0, VGB.size);

	//��������SVO���飬�ӵڶ�����ʼ
	uint32_t svoDepth = 2;	//���SVO����ȣ���1��ʼ�������ڵ�Ϊ��һ��
	uint32_t vgmSize = 2;
	while (vgmSize < setting.voxelNum) {
		svoDepth++;
		vgmSize <<= 1;
	}

	this->SVONodeBlockInfos.resize(0);
	this->SVONodeTempInfos.resize(0);
	this->SVONodeCount.resize(svoDepth - 2);
	this->SVOs_PG.resize(svoDepth - 2);	 //���洢���ڵ��Ҷ�ڵ�
	for (int i = 0; i < svoDepth - 2; ++i) {
		uint32_t nodeCount = std::pow(8, i + 1);
		CHECK(hipMalloc((void**)&this->SVOs_PG[i], nodeCount * sizeof(FzbSVONodeData_PG)));
		uint32_t blockSize = nodeCount > 1024 ? 1024 : nodeCount;
		uint32_t gridSize = (nodeCount + blockSize - 1) / blockSize;
		initSVO << <gridSize, blockSize >> > (this->SVOs_PG[i], nodeCount);

		CHECK(hipMalloc((void**)&SVONodeCount[i], sizeof(uint32_t)));
		CHECK(hipMemset(SVONodeCount[i], 0, sizeof(uint32_t)));

		if (nodeCount <= createSVOKernelBlockSize) continue;
		uint32_t blockCount = nodeCount / createSVOKernelBlockSize;
		FzbSVONodeBlock* blockInfo;
		CHECK(hipMalloc((void**)&blockInfo, blockCount * sizeof(FzbSVONodeBlock)));
		CHECK(hipMemset(blockInfo, 0, blockCount * sizeof(FzbSVONodeBlock)));
		this->SVONodeBlockInfos.push_back(blockInfo);

		FzbSVONodeTempInfo* nodeTempInfo;
		CHECK(hipMalloc((void**)&nodeTempInfo, nodeCount * sizeof(FzbSVONodeTempInfo)));
		CHECK(hipMemset(nodeTempInfo, 0, nodeCount * sizeof(FzbSVONodeTempInfo)));
		this->SVONodeTempInfos.push_back(nodeTempInfo);
	}

	this->extSvoSemaphore_PG = importVulkanSemaphoreObjectFromNTHandle(SVOFinishedSemaphore_PG);

	//����cuda���ã������ʹ��L1 cache
	hipFuncSetAttribute(reinterpret_cast<const void*>(lightInject_cuda), hipFuncAttributeMaxDynamicSharedMemorySize, 0);
}

//--------------------------------------------------------------------��ʼ��VGB-------------------------------------------------------------------------
__global__ void initVGB_Cuda(FzbVoxelData_PG* VGB, uint32_t voxelCount) {
	uint32_t threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
	if (threadIndex >= voxelCount) return;
	FzbVoxelData_PG data;
	data.hasData = 0;
	data.AABB.leftX = __float_as_int(FLT_MAX);
	data.AABB.leftY = __float_as_int(FLT_MAX);
	data.AABB.leftZ = __float_as_int(FLT_MAX);
	data.AABB.rightX = __float_as_int(-FLT_MAX);
	data.AABB.rightY = __float_as_int(-FLT_MAX);
	data.AABB.rightZ = __float_as_int(-FLT_MAX);
	data.irradiance = glm::vec3(0.0f);
	VGB[threadIndex] = data;
}
void FzbSVOCuda_PG::initVGB() {
	uint32_t voxelCount = std::pow(setting.voxelNum, 3);
	uint32_t gridSize = (voxelCount + 1023) / 1024;
	initVGB_Cuda << <gridSize, 1024 >> > (VGB, voxelCount);
	CHECK(hipDeviceSynchronize());
}
//--------------------------------------------------------------------����ע��-------------------------------------------------------------------------
__device__ int getVGBVoxelIndex(int voxelCount, glm::ivec3& voxelIndex) {
	int voxelTotalCount = voxelCount * voxelCount * voxelCount;
	int voxelIndexU = 0;
	while (voxelTotalCount > 1) {
		voxelCount = voxelCount / 2;
		voxelTotalCount = voxelTotalCount / 8;
		if (voxelIndex.z / voxelCount == 1) {
			voxelIndexU += 4 * voxelTotalCount;
			voxelIndex.z -= voxelCount;
		}
		if (voxelIndex.y / voxelCount == 1) {
			voxelIndexU += 2 * voxelTotalCount;
			voxelIndex.y -= voxelCount;
		}
		if (voxelIndex.x / voxelCount == 1) {
			voxelIndexU += voxelTotalCount;
			voxelIndex.x -= voxelCount;
		}
	}
	return voxelIndexU;
}
__device__ void lightInject_getRadiance(FzbTriangleAttribute& triangleAttribute, FzbRay& ray, const FzbRayTracingLightSet* lightSet,
	const float* __restrict__ vertices, const hipTextureObject_t* __restrict__ materialTextures,
	const FzbBvhNode* __restrict__ bvhNodeArray, const FzbBvhNodeTriangleInfo* __restrict__ bvhTriangleInfoArray, uint32_t& randomNumberSeed,
	glm::vec3& irradiance, glm::vec3& radiance) {
	irradiance = triangleAttribute.emissive;
	radiance = triangleAttribute.emissive;
	FzbRay tempRay;
	FzbTriangleAttribute hitTriangleAttribute;
	for (int i = 0; i < lightSet->pointLightCount; ++i) {
		const FzbRayTracingPointLight& light = lightSet->pointLightInfoArray[i];
		glm::vec3 direction = light.worldPos - ray.hitPos;
		if (glm::dot(direction, triangleAttribute.normal) <= 0) continue;
		tempRay.depth = FLT_MAX;
		tempRay.direction = glm::normalize(direction);
		bool hit = sceneCollisionDetection(bvhNodeArray, bvhTriangleInfoArray, vertices, materialTextures, tempRay, hitTriangleAttribute, false);
		if (!hit) continue;
		else if (abs(tempRay.depth - glm::length(direction)) > 0.1f) continue;
		float r2 = glm::length(direction); r2 *= r2;
		float cosTheta = glm::clamp(glm::dot(triangleAttribute.normal, tempRay.direction), 0.0f, 1.0f);
		glm::vec3 irradiance_temp = cosTheta * light.radiantIntensity / r2;
		irradiance += irradiance_temp;
		radiance += irradiance_temp * getBSDF(triangleAttribute, tempRay.direction, -ray.direction, ray);
	}
	for (int i = 0; i < lightSet->areaLightCount; ++i) {
		const FzbRayTracingAreaLight& light = lightSet->areaLightInfoArray[i];
		float randomNumberX = rand(randomNumberSeed);
		float randomNumberY = rand(randomNumberSeed);
		glm::vec3 lightPos = glm::vec3(light.worldPos + randomNumberX * light.edge0 + randomNumberY * light.edge1);
		glm::vec3 direction = lightPos - ray.hitPos;
		if (triangleAttribute.materialType != 2 && glm::dot(direction, triangleAttribute.normal) <= 0) continue;
		tempRay.startPos = ray.hitPos + direction * 0.001f;
		tempRay.depth = FLT_MAX;
		tempRay.direction = glm::normalize(direction);
		float r = glm::length(direction);
		bool hit = sceneCollisionDetection(bvhNodeArray, bvhTriangleInfoArray, vertices, materialTextures, tempRay, hitTriangleAttribute, false);
		if (!hit) continue;
		else if (abs(tempRay.depth - r) > 0.1f) continue;
		glm::vec3 lightRadiance_cosTheta = light.radiance * glm::clamp(glm::dot(triangleAttribute.normal, tempRay.direction), 0.0f, 1.0f);
		lightRadiance_cosTheta *= light.area;	// pdf = 1 / area
		lightRadiance_cosTheta *= glm::clamp(glm::dot(-light.normal, tempRay.direction), 0.0f, 1.0f);	//΢�ֵ�λ��dw��ΪdA
		r = glm::max(r, 1.0f);
		glm::vec3 irradiance_temp = lightRadiance_cosTheta / (r * r);
		irradiance += irradiance_temp;
		radiance += irradiance_temp * getBSDF(triangleAttribute, tempRay.direction, -ray.direction, tempRay);
	}
}
__global__ void lightInject_cuda(FzbVoxelData_PG* VGB, const float* __restrict__ vertices, const hipTextureObject_t* __restrict__ materialTextures,
	const FzbBvhNode* __restrict__ bvhNodeArray, const FzbBvhNodeTriangleInfo* __restrict__ bvhTriangleInfoArray, const uint32_t rayCount) {
	__shared__ FzbPathTracingCameraInfo groupCameraInfo;				//216B
	__shared__ FzbVGBUniformData groupVGBUniformData;				//216B
	__shared__ uint32_t groupRandomNumberSeed;
	__shared__ FzbRayTracingPointLight groupPointLightInfoArray[maxPointLightCount];	//512B
	__shared__ FzbRayTracingAreaLight grouprAreaLightInfoArray[maxAreaLightCount];		//692B
	__shared__ FzbRayTracingLightSet lightSet;

	volatile const uint32_t threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
	if (threadIndex >= rayCount) return;
	if (threadIdx.x < systemPointLightCount) groupPointLightInfoArray[threadIdx.x] = systemPointLightInfoArray[threadIdx.x];
	if (threadIdx.x < systemAreaLightCount) grouprAreaLightInfoArray[threadIdx.x] = systemAreaLightInfoArray[threadIdx.x];
	if (threadIdx.x == 0) {
		groupCameraInfo = systemCameraInfo;
		groupVGBUniformData = systemVGBUniformData;
		groupRandomNumberSeed = systemRandomNumberSeed;
		lightSet.pointLightCount = systemPointLightCount;
		lightSet.areaLightCount = systemAreaLightCount;
		lightSet.pointLightInfoArray = groupPointLightInfoArray;
		lightSet.areaLightInfoArray = grouprAreaLightInfoArray;
	}
	__syncthreads();

	uint32_t randomNumberSeed = groupRandomNumberSeed + threadIndex;
	uint2 seed2 = pcg2d(make_uint2(threadIndex) * randomNumberSeed);
	randomNumberSeed = seed2.x + seed2.y;

	const uint32_t maxPathDepth = 3;
	glm::vec3 voxelRadiance[maxPathDepth];	//������һ��ײ�����radiance
	glm::vec3 voxelIrradiances[maxPathDepth];	//��ǰײ����õ���irradiance������NEE + ��һ�����radiance
	uint32_t voxelIndices[maxPathDepth];
	glm::vec3 voxelBSDF[maxPathDepth - 1];
	float voxelPDF[maxPathDepth - 1];
	float voxelCosTheta[maxPathDepth - 1];

	float RR = 0.8f;
	bool hit = true;
	FzbTriangleAttribute hitTriangleAttribute;
	FzbTriangleAttribute lastHitTriangleAttribute;

	glm::vec2 texelXY = glm::vec2(threadIndex % groupCameraInfo.screenWidth, threadIndex / groupCameraInfo.screenWidth);
	glm::vec4 screenPos = glm::vec4(((texelXY + glm::vec2(0.5f)) / glm::vec2(groupCameraInfo.screenWidth, groupCameraInfo.screenHeight)) * 2.0f - 1.0f, 0.0f, 1.0f);	//vulkan�н�ƽ��ndcDepth��[0,1]
	screenPos = groupCameraInfo.inversePVMatrix * screenPos;
	screenPos /= screenPos.w;
	FzbRay ray;
	ray.startPos = groupCameraInfo.cameraWorldPos;
	ray.direction = glm::normalize(glm::vec3(screenPos) - ray.startPos);
	ray.depth = FLT_MAX;
	ray.refraction = false;
	ray.ext = true;
	glm::vec3 lastDirection = -ray.direction;

	hit = sceneCollisionDetection(bvhNodeArray, bvhTriangleInfoArray, vertices, materialTextures, ray, hitTriangleAttribute);
	if (!hit) return;
	glm::ivec3 voxelIndex = glm::ivec3((ray.hitPos - groupVGBUniformData.voxelStartPos) / groupVGBUniformData.voxelSize);
	voxelIndices[0] = getVGBVoxelIndex(groupVGBUniformData.voxelCount, voxelIndex);
	lightInject_getRadiance(hitTriangleAttribute, ray, &lightSet,
		vertices, materialTextures, bvhNodeArray, bvhTriangleInfoArray, randomNumberSeed,
		voxelIrradiances[0], voxelRadiance[0]);

	int pathLength = 1;
#pragma nounroll
	while (pathLength < maxPathDepth) {
		float randomNumber = rand(randomNumberSeed);
		if (randomNumber > RR) break;
		voxelPDF[pathLength - 1] = RR;

		lastDirection = -ray.direction;
		generateRay(hitTriangleAttribute, voxelPDF[pathLength - 1], ray, randomNumberSeed);

		lastHitTriangleAttribute = hitTriangleAttribute;
		hit = sceneCollisionDetection(bvhNodeArray, bvhTriangleInfoArray, vertices, materialTextures, ray, hitTriangleAttribute);
		if (!hit) break;

		voxelBSDF[pathLength - 1] = getBSDF(lastHitTriangleAttribute, ray.direction, lastDirection, ray);
		voxelCosTheta[pathLength - 1] = glm::abs(glm::dot(ray.direction, lastHitTriangleAttribute.normal));

		voxelIndex = glm::ivec3((ray.hitPos - groupVGBUniformData.voxelStartPos) / groupVGBUniformData.voxelSize);
		voxelIndices[pathLength] = getVGBVoxelIndex(groupVGBUniformData.voxelCount, voxelIndex);
		lightInject_getRadiance(hitTriangleAttribute, ray, &lightSet,
			vertices, materialTextures, bvhNodeArray, bvhTriangleInfoArray, randomNumberSeed,
			voxelIrradiances[pathLength], voxelRadiance[pathLength]);

		++pathLength;
	}

	atomicAdd(&VGB[voxelIndices[pathLength - 1]].irradiance.x, voxelIrradiances[pathLength - 1].x);
	atomicAdd(&VGB[voxelIndices[pathLength - 1]].irradiance.y, voxelIrradiances[pathLength - 1].y);
	atomicAdd(&VGB[voxelIndices[pathLength - 1]].irradiance.z, voxelIrradiances[pathLength - 1].z);
	glm::vec3 radiance = voxelRadiance[pathLength - 1];
	for (int i = pathLength - 2; i >= 0; --i) {
		voxelIrradiances[i] += radiance * voxelCosTheta[i];

		atomicAdd(&VGB[voxelIndices[i]].irradiance.x, voxelIrradiances[i].x);
		atomicAdd(&VGB[voxelIndices[i]].irradiance.y, voxelIrradiances[i].y);
		atomicAdd(&VGB[voxelIndices[i]].irradiance.z, voxelIrradiances[i].z);

		radiance = voxelRadiance[i] + radiance * voxelBSDF[i] / voxelPDF[i];
	}
}

void FzbSVOCuda_PG::lightInject() {
	this->sourceManager->createRuntimeSource();

	VkExtent2D resolution = FzbRenderer::globalData.getResolution();
	uint32_t texelCount = resolution.width * resolution.height;
	uint32_t rayCount = texelCount;
	uint32_t blockSize = 512;
	uint32_t gridSize = (rayCount + blockSize - 1) / blockSize;

	lightInject_cuda<<< gridSize, blockSize, 0, sourceManager->stream>>> (VGB, sourceManager->vertices, sourceManager->materialTextures, sourceManager->bvhNodeArray, sourceManager->bvhTriangleInfoArray, rayCount);
}
//--------------------------------------------------------------------����SVO_PG-------------------------------------------------------------------------
__global__ void createSVO_PG_device_first(const FzbVoxelData_PG* __restrict__ VGB, FzbSVONodeData_PG* SVONodes, uint32_t voxelCount) {
	__shared__ FzbVGBUniformData groupVGBUniformData;
	__shared__ FzbSVOUnformData groupSVOUniformData;

	uint32_t threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t warpIndex = threadIdx.x / 32;
	uint32_t warpLane = threadIdx.x & 31;
	if (threadIndex >= voxelCount) return;
	if (threadIdx.x == 0) {
		groupVGBUniformData = systemVGBUniformData;
		groupSVOUniformData = systemSVOUniformData;
	}
	__syncthreads();

	//�����blockָ���Ǹ���node
	uint32_t indexInBlock = threadIndex & 7;	//��8���ֵ�node�е�����
	uint32_t blockIndex = threadIndex / 8;		//block��ȫ�ֵ�����
	uint32_t blockIndexInWarpBit = (blockIndex & 3) * 8;	//��ǰblock��warp�е�λ����
	uint32_t blockIndexInGroup = threadIdx.x / 8;

	FzbVoxelData_PG voxelData = VGB[threadIndex];
	bool hasData = voxelData.hasData && voxelData.irradiance != glm::vec3(0.0f);
	uint32_t activeMask = __ballot_sync(0xFFFFFFFF, hasData);
	int firstActiveLaneInBlock = __ffs(activeMask & (0xff << blockIndexInWarpBit)) - 1;
	if (firstActiveLaneInBlock == -1) return;	//��ǰblock��nodeȫ��û������

	FzbAABB AABB = { FLT_MAX, -FLT_MAX, FLT_MAX, -FLT_MAX, FLT_MAX, -FLT_MAX };
	if (hasData) {
		AABB = {
			__int_as_float(voxelData.AABB.leftX),
			__int_as_float(voxelData.AABB.rightX),
			__int_as_float(voxelData.AABB.leftY),
			__int_as_float(voxelData.AABB.rightY),
			__int_as_float(voxelData.AABB.leftZ),
			__int_as_float(voxelData.AABB.rightZ)
		};
	}
	if (__popc(activeMask) == 1) {	//ֻ��һ����ֵ��node����ֱ�Ӹ�ֵ����
		if (hasData) {
			SVONodes[blockIndex].indivisible = 1;
			SVONodes[blockIndex].AABB = AABB;
			SVONodes[blockIndex].irradiance = voxelData.irradiance;
		}
		return;
	}
	//------------------------------------------------irradiance�ж�-------------------------------------------------
	uint indivisible = 1;
	float irrdianceValue = glm::length(voxelData.irradiance);
	uint32_t ignore = 0;
	for (int i = 0; i < 8; ++i) {
		float other_val = __shfl_sync(0xFFFFFFFF, irrdianceValue, blockIndexInWarpBit + i);
		float minIrradiance = min(irrdianceValue, other_val);
		float maxIrradiance = max(irrdianceValue, other_val);
		if (minIrradiance == 0.0f) continue;
		if (maxIrradiance / minIrradiance > groupSVOUniformData.irradianceThreshold) {
			if (irrdianceValue == minIrradiance) {
				if (irrdianceValue < groupSVOUniformData.ignoreIrradianceValueThreshold) ignore = 1;
				else indivisible = 0;
			}
		}
	}
	for (int offset = 4; offset > 0; offset /= 2) {
		uint32_t other_val = __shfl_down_sync(0xFFFFFFFF, indivisible, offset);
		indivisible = indivisible & other_val;
	}
	//------------------------------------------------����irradiance-------------------------------------------------
	glm::vec3 mergeIrradianceTotal = voxelData.irradiance;
	for (int offset = 4; offset > 0; offset /= 2) {
		mergeIrradianceTotal.x += __shfl_down_sync(0xFFFFFFFF, mergeIrradianceTotal.x, offset);
		mergeIrradianceTotal.y += __shfl_down_sync(0xFFFFFFFF, mergeIrradianceTotal.y, offset);
		mergeIrradianceTotal.z += __shfl_down_sync(0xFFFFFFFF, mergeIrradianceTotal.z, offset);
	}
	glm::vec3 mergeIrradiance = ignore ? glm::vec3(0.0f) : voxelData.irradiance;
	for (int offset = 4; offset > 0; offset /= 2) {
		mergeIrradiance.x += __shfl_down_sync(0xFFFFFFFF, mergeIrradiance.x, offset);
		mergeIrradiance.y += __shfl_down_sync(0xFFFFFFFF, mergeIrradiance.y, offset);
		mergeIrradiance.z += __shfl_down_sync(0xFFFFFFFF, mergeIrradiance.z, offset);
	}
	//------------------------------------------�õ����Ϻ��AABB---------------------------------------------------------
	if (ignore == 1) AABB = { FLT_MAX, -FLT_MAX, FLT_MAX, -FLT_MAX, FLT_MAX, -FLT_MAX };
	FzbAABB mergeAABB = AABB;
	//�õ����Ϻ��AABB��left
	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.leftX, offset);
		mergeAABB.leftX = fminf(mergeAABB.leftX, other_val);
	}
	mergeAABB.leftX = __shfl_sync(0xFFFFFFFF, mergeAABB.leftX, blockIndexInWarpBit);

	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.leftY, offset);
		mergeAABB.leftY = fminf(mergeAABB.leftY, other_val);
	}
	mergeAABB.leftY = __shfl_sync(0xFFFFFFFF, mergeAABB.leftY, blockIndexInWarpBit);

	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.leftZ, offset);
		mergeAABB.leftZ = fminf(mergeAABB.leftZ, other_val);
	}
	mergeAABB.leftZ = __shfl_sync(0xFFFFFFFF, mergeAABB.leftZ, blockIndexInWarpBit);

	//�õ����Ϻ��AABB��right
	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.rightX, offset);
		mergeAABB.rightX = fmaxf(mergeAABB.rightX, other_val);
	}
	mergeAABB.rightX = __shfl_sync(0xFFFFFFFF, mergeAABB.rightX, blockIndexInWarpBit);

	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.rightY, offset);
		mergeAABB.rightY = fmaxf(mergeAABB.rightY, other_val);
	}
	mergeAABB.rightY = __shfl_sync(0xFFFFFFFF, mergeAABB.rightY, blockIndexInWarpBit);

	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.rightZ, offset);
		mergeAABB.rightZ = fmaxf(mergeAABB.rightZ, other_val);
	}
	mergeAABB.rightZ = __shfl_sync(0xFFFFFFFF, mergeAABB.rightZ, blockIndexInWarpBit);
	//------------------------------------------------��������-------------------------------------------------
	float surfaceArea = 0.0f;
	if (hasData && ignore == 0) {
		float lengthX = AABB.rightX - AABB.leftX;
		float lengthY = AABB.rightY - AABB.leftY;
		float lengthZ = AABB.rightZ - AABB.leftZ;
		surfaceArea = (lengthX * lengthY + lengthX * lengthZ + lengthY * lengthZ) * 2;
	}
	for (int offset = 4; offset > 0; offset /= 2) {
		surfaceArea += __shfl_down_sync(0xFFFFFFFF, surfaceArea, offset);
	}
	//--------------------------------------------------�Ը��ڵ㸳ֵ-------------------------------------------------
	if (warpLane == blockIndexInWarpBit) {
		float lengthX = mergeAABB.rightX - mergeAABB.leftX;
		float lengthY = mergeAABB.rightY - mergeAABB.leftY;
		float lengthZ = mergeAABB.rightZ - mergeAABB.leftZ;
		float mergeSurfaceArea = (lengthX * lengthY + lengthX * lengthZ + lengthY * lengthZ) * 2;
		if (surfaceArea != 0.0f && mergeSurfaceArea / surfaceArea > groupSVOUniformData.surfaceAreaThreshold) indivisible = 0;
	
		SVONodes[blockIndex].indivisible = indivisible;
		if (indivisible) SVONodes[blockIndex].pdf = glm::length(mergeIrradiance) / glm::length(mergeIrradianceTotal);
		SVONodes[blockIndex].irradiance = mergeIrradianceTotal;
		SVONodes[blockIndex].AABB = mergeAABB;
	}
}
__global__ void createSVO_PG_device(FzbSVONodeData_PG* SVONodes_children, FzbSVONodeData_PG* SVONodes, uint32_t nodeCount) {
	__shared__ FzbSVOUnformData groupSVOUniformData;

	uint32_t threadIndex = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t warpIndex = threadIdx.x / 32;
	uint32_t warpLane = threadIdx.x & 31;
	if (threadIndex >= nodeCount * nodeCount * nodeCount) return;
	if (threadIdx.x == 0) {
		groupSVOUniformData = systemSVOUniformData;
	}
	__syncthreads();

	//�����blockָ���Ǹ���node
	uint32_t indexInBlock = threadIndex & 7;	//��8���ֵ�node�е�����
	uint32_t blockIndex = threadIndex / 8;		//block��ȫ�ֵ�����
	uint32_t blockIndexInWarpBit = (blockIndex & 3) * 8;	//��ǰblock��warp�е�λ����
	uint32_t blockIndexInGroup = threadIdx.x / 8;
	//uint32_t blockCount = nodeCount / 2;	//ÿ�����м���block
	//uint32_t nodeIndexZ = (blockIndex / (blockCount * blockCount));
	//uint32_t nodeIndexY = (blockIndex - nodeIndexZ * (blockCount * blockCount)) / blockCount;
	//uint32_t nodeIndexX = blockIndex % blockCount;
	//nodeIndexX = nodeIndexX * 2 + (indexInBlock & 1);
	//nodeIndexY = nodeIndexY * 2 + ((indexInBlock >> 1) & 1);
	//nodeIndexZ = nodeIndexZ * 2 + ((indexInBlock >> 2) & 1);
	//uint32_t voxelIndexU = nodeIndexZ * (nodeCount * nodeCount) +
	//	nodeIndexY * nodeCount + nodeIndexX;
	FzbSVONodeData_PG nodeData = SVONodes_children[threadIndex];
	bool hasData = glm::length(nodeData.irradiance) > 0.01f;
	uint32_t activeMask = __ballot_sync(0xFFFFFFFF, hasData);
	int firstActiveLaneInBlock = __ffs(activeMask & (0xff << blockIndexInWarpBit)) - 1;
	if (firstActiveLaneInBlock == -1) return;	//��ǰblock��nodeȫ��û������

	if (__popc(activeMask) == 1) {	//ֻ��һ����ֵ��node����ֱ�Ӹ�ֵ����
		if (hasData) {
			SVONodes[blockIndex].indivisible = 1;
			SVONodes[blockIndex].AABB = nodeData.AABB;
			SVONodes[blockIndex].irradiance = nodeData.irradiance;
		}
		return;
	}
	//------------------------------------------------irradiance�ж�-------------------------------------------------
	uint indivisible = 1;
	float irrdianceValue = glm::length(nodeData.irradiance);
	uint32_t ignore = 0;
	for (int i = 0; i < 8; ++i) {
		float other_val = __shfl_sync(0xFFFFFFFF, irrdianceValue, blockIndexInWarpBit + i);
		float minIrradiance = min(irrdianceValue, other_val);
		float maxIrradiance = max(irrdianceValue, other_val);
		if (minIrradiance == 0.0f) continue;
		if (maxIrradiance / minIrradiance > groupSVOUniformData.irradianceThreshold) {
			if (irrdianceValue == minIrradiance) {
				if (irrdianceValue < groupSVOUniformData.ignoreIrradianceValueThreshold) ignore = 1;
				else indivisible = 0;
			}
		}
	}
	for (int offset = 4; offset > 0; offset /= 2) {
		uint32_t other_val = __shfl_down_sync(0xFFFFFFFF, indivisible, offset);
		indivisible = indivisible & other_val;
	}
	//------------------------------------------------����irradiance-------------------------------------------------
	glm::vec3 mergeIrradianceTotal = nodeData.irradiance;
	for (int offset = 4; offset > 0; offset /= 2) {
		mergeIrradianceTotal.x += __shfl_down_sync(0xFFFFFFFF, mergeIrradianceTotal.x, offset);
		mergeIrradianceTotal.y += __shfl_down_sync(0xFFFFFFFF, mergeIrradianceTotal.y, offset);
		mergeIrradianceTotal.z += __shfl_down_sync(0xFFFFFFFF, mergeIrradianceTotal.z, offset);
	}
	glm::vec3 mergeIrradiance = ignore ? glm::vec3(0.0f) : nodeData.irradiance;
	for (int offset = 4; offset > 0; offset /= 2) {
		mergeIrradiance.x += __shfl_down_sync(0xFFFFFFFF, mergeIrradiance.x, offset) * __shfl_down_sync(0xFFFFFFFF, nodeData.pdf, offset);
		mergeIrradiance.y += __shfl_down_sync(0xFFFFFFFF, mergeIrradiance.y, offset) * __shfl_down_sync(0xFFFFFFFF, nodeData.pdf, offset);
		mergeIrradiance.z += __shfl_down_sync(0xFFFFFFFF, mergeIrradiance.z, offset) * __shfl_down_sync(0xFFFFFFFF, nodeData.pdf, offset);
	}
	//------------------------------------------�õ����Ϻ��AABB---------------------------------------------------------
	if (ignore == 1) nodeData.AABB = { FLT_MAX, -FLT_MAX, FLT_MAX, -FLT_MAX, FLT_MAX, -FLT_MAX };
	FzbAABB mergeAABB = nodeData.AABB;
	//�õ����Ϻ��AABB��left
	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.leftX, offset);
		mergeAABB.leftX = fminf(mergeAABB.leftX, other_val);
	}
	mergeAABB.leftX = __shfl_sync(0xFFFFFFFF, mergeAABB.leftX, blockIndexInWarpBit);

	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.leftY, offset);
		mergeAABB.leftY = fminf(mergeAABB.leftY, other_val);
	}
	mergeAABB.leftY = __shfl_sync(0xFFFFFFFF, mergeAABB.leftY, blockIndexInWarpBit);

	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.leftZ, offset);
		mergeAABB.leftZ = fminf(mergeAABB.leftZ, other_val);
	}
	mergeAABB.leftZ = __shfl_sync(0xFFFFFFFF, mergeAABB.leftZ, blockIndexInWarpBit);

	//�õ����Ϻ��AABB��right
	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.rightX, offset);
		mergeAABB.rightX = fmaxf(mergeAABB.rightX, other_val);
	}
	mergeAABB.rightX = __shfl_sync(0xFFFFFFFF, mergeAABB.rightX, blockIndexInWarpBit);

	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.rightY, offset);
		mergeAABB.rightY = fmaxf(mergeAABB.rightY, other_val);
	}
	mergeAABB.rightY = __shfl_sync(0xFFFFFFFF, mergeAABB.rightY, blockIndexInWarpBit);

	for (int offset = 4; offset > 0; offset /= 2) {
		float other_val = __shfl_down_sync(0xFFFFFFFF, mergeAABB.rightZ, offset);
		mergeAABB.rightZ = fmaxf(mergeAABB.rightZ, other_val);
	}
	mergeAABB.rightZ = __shfl_sync(0xFFFFFFFF, mergeAABB.rightZ, blockIndexInWarpBit);
	//------------------------------------------------��������-------------------------------------------------
	float surfaceArea = 0.0f;
	if (hasData) {
		float lengthX = nodeData.AABB.rightX - nodeData.AABB.leftX;
		float lengthY = nodeData.AABB.rightY - nodeData.AABB.leftY;
		float lengthZ = nodeData.AABB.rightZ - nodeData.AABB.leftZ;
		surfaceArea = (lengthX * lengthY + lengthX * lengthZ + lengthY * lengthZ) * 2;
	}
	for (int offset = 4; offset > 0; offset /= 2) {
		surfaceArea += __shfl_down_sync(0xFFFFFFFF, surfaceArea, offset);
	}
	//--------------------------------------------------�Ը��ڵ㸳ֵ-------------------------------------------------
	if (warpLane == blockIndexInWarpBit) {
		float lengthX = mergeAABB.rightX - mergeAABB.leftX;
		float lengthY = mergeAABB.rightY - mergeAABB.leftY;
		float lengthZ = mergeAABB.rightZ - mergeAABB.leftZ;
		float mergeSurfaceArea = (lengthX * lengthY + lengthX * lengthZ + lengthY * lengthZ) * 2;
		if (surfaceArea != 0.0f && mergeSurfaceArea / surfaceArea > groupSVOUniformData.surfaceAreaThreshold) indivisible = 0;

		SVONodes[blockIndex].indivisible = indivisible;
		if (indivisible) SVONodes[blockIndex].pdf = glm::length(mergeIrradiance) / glm::length(mergeIrradianceTotal);
		SVONodes[blockIndex].irradiance = mergeIrradianceTotal;
		SVONodes[blockIndex].AABB = mergeAABB;
	}
}

template<bool notOnlyOneBlock>
__global__ void compressSVO_PG_firstStep(FzbSVONodeData_PG* SVONodes, FzbSVONodeBlock* blockInfo,
	FzbSVONodeTempInfo* SVONodeTempInfos, uint32_t* SVONodeCount, uint32_t nodeCount) {
	__shared__ uint64_t groupHasDataNodeBlockMask;	//һ���߳���512��node��ÿ��8��nodeΪһ�飬����64�飬groupHasDataNodeBlockÿһλ��ʾһ������ֵ
	__shared__ uint32_t groupHasDataNodeCountInWarp[16];	//ÿ��warp����ֵnode������
	uint32_t threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadIndex >= nodeCount) return;
	uint32_t blockIndexInGroup = threadIdx.x / 8;
	uint32_t laneInBlock = threadIdx.x & 7;
	uint32_t warpIndex = threadIdx.x / 32;
	uint32_t warpLane = threadIdx.x & 31;
	uint32_t firstBlockLaneInWarp = (blockIndexInGroup & 3) * 8;
	if (threadIdx.x == 0) groupHasDataNodeBlockMask = 0;
	if (threadIdx.x < 16) groupHasDataNodeCountInWarp[threadIdx.x] = 0;
	__syncthreads();

	bool hasData = glm::length(SVONodes[threadIndex].irradiance) != 0.0f;
	bool blockHasData = hasData;
	for (int offset = 4; offset > 0; offset /= 2) 	//ֻҪһ��node��ֵ�����nodeBlock����ֵ
		blockHasData |= __shfl_down_sync(0xFFFFFFFF, blockHasData, offset);

	uint32_t warpNodeMask = hasData << warpLane;	//���warp����ֵnode��mask
	for (int offset = 16; offset > 0; offset /= 2)
		warpNodeMask |= __shfl_down_sync(0xFFFFFFFF, warpNodeMask, offset);

	uint8_t warpHasDataBlockMask = blockHasData;	//���warp����ֵnodeBlock��mask
	warpHasDataBlockMask |= __shfl_sync(0xFFFFFFFF, blockHasData, 8) << 1;
	warpHasDataBlockMask |= __shfl_sync(0xFFFFFFFF, blockHasData, 16) << 2;
	warpHasDataBlockMask |= __shfl_sync(0xFFFFFFFF, blockHasData, 24) << 3;
	if (warpLane == 0) {
		atomicOr(&groupHasDataNodeBlockMask, warpHasDataBlockMask << (warpIndex * 4));
		groupHasDataNodeCountInWarp[warpIndex] = __popc(warpNodeMask);
	}
	warpNodeMask = __shfl_sync(0xFFFFFFFF, warpNodeMask, 0);
	__syncthreads();

	uint32_t label = 0;
	if (warpLane == 0) {
		for (int i = 0; i < warpIndex; ++i) label += groupHasDataNodeCountInWarp[i];	//֪����ǰ�߳�����ǰ��warp����ֵnode����
	}
	label = __shfl_sync(0xFFFFFFFF, label, 0);
	label += __popc(warpNodeMask << (32 - warpLane));	//����warp��ǰ���м�����ֵnode���õ�����������߳������ǵڼ�����ֵnode

	uint32_t nodeIndex = __popcll(groupHasDataNodeBlockMask << blockIndexInGroup) * 8;
	nodeIndex += laneInBlock;
	if (threadIdx.x == 0) atomicAdd(SVONodeCount, __popcll(groupHasDataNodeBlockMask) * 8);

	if constexpr (notOnlyOneBlock) {
		FzbSVONodeTempInfo tempInfo;
		tempInfo.nodeData = SVONodes[threadIndex];;
		tempInfo.nodeIndexInThreadBlock = nodeIndex;
		SVONodeTempInfos[threadIndex] = tempInfo;
		if (hasData) SVONodeTempInfos[threadIndex].nodeData.label = label;

		if (warpIndex == 0) {
			uint32_t blockHasDataNodeTotalCount = threadIdx.x < 16 ? groupHasDataNodeCountInWarp[threadIdx.x] : 0;
			for (int offset = 16; offset > 0; offset /= 2)
				blockHasDataNodeTotalCount += __shfl_down_sync(0xFFFFFFFF, blockHasDataNodeTotalCount, offset);
			if (threadIdx.x == 0) {
				blockInfo[blockIdx.x].nodeCount = blockHasDataNodeTotalCount;
				blockInfo[blockIdx.x].blockCount = __popcll(groupHasDataNodeBlockMask);
			}
		}
	}
	if constexpr (!notOnlyOneBlock) {
		if (hasData) {
			FzbSVONodeData_PG nodeData = SVONodes[threadIndex];
			nodeData.label = label;
			SVONodes[nodeIndex] = nodeData;
		}
	}
}
__global__ void compressSVO_PG_secondStep(FzbSVONodeData_PG* SVONodes, FzbSVONodeBlock* blockInfo,
	FzbSVONodeTempInfo* SVONodeTempInfos, uint32_t nodeCount) {
	__shared__ uint32_t groupLabel;
	__shared__ uint32_t groupBlockStartIndex;
	
	uint32_t threadIndex = blockDim.x * blockIdx.x + threadIdx.x;
	if (threadIndex >= nodeCount) return;
	if (threadIdx.x == 0) groupLabel = 0;
	__syncthreads();
	uint32_t warpIndex = threadIdx.x / 32;
	uint32_t warpLane = threadIdx.x & 31;

	bool hasData = glm::length(SVONodeTempInfos[threadIndex].nodeData.irradiance) != 0.0f;
	uint32_t label = 0;
	if (warpIndex == 0) {
		label = threadIdx.x < blockIdx.x ? blockInfo[threadIdx.x].nodeCount : 0;
		uint32_t blockStartIndex = threadIdx.x < blockIdx.x ? blockInfo[threadIdx.x].blockCount : 0;
		for (int offset = 16; offset > 0; offset /= 2) {
			label += __shfl_down_sync(0xFFFFFFFF, label, offset);
			blockStartIndex += __shfl_down_sync(0xFFFFFFFF, blockStartIndex, offset);
		}
		if (threadIdx.x == 0) {
			groupLabel = label;
			groupBlockStartIndex = blockStartIndex;
		}
	}
	__syncthreads();
	label = groupLabel;
	uint32_t nodeIndex = groupBlockStartIndex + SVONodeTempInfos[threadIndex].nodeIndexInThreadBlock;

	if (hasData) {
		FzbSVONodeData_PG nodeData = SVONodeTempInfos[threadIndex].nodeData;
		nodeData.label += label;
		SVONodes[nodeIndex] = nodeData;
	}
}

void FzbSVOCuda_PG::createSVOCuda_PG() {
	uint32_t voxelCount = std::pow(setting.voxelNum, 3);
	uint32_t blockSize = createSVOKernelBlockSize;
	uint32_t gridSize = (voxelCount + blockSize - 1) / blockSize;
	createSVO_PG_device_first << <gridSize, blockSize, 0, sourceManager->stream >> > (VGB, SVOs_PG[SVOs_PG.size() - 1], voxelCount);
	for (int i = SVOs_PG.size() - 1; i > 0; --i) {
		FzbSVONodeData_PG* SVONodes_children = SVOs_PG[i];
		FzbSVONodeData_PG* SVONodes = SVOs_PG[i - 1];
		uint32_t nodeCount = setting.voxelNum / pow(2, SVOs_PG.size() - i);
		uint32_t nodeTotalCount = nodeCount * nodeCount * nodeCount;
		blockSize = nodeTotalCount > createSVOKernelBlockSize ? createSVOKernelBlockSize : nodeTotalCount;
		gridSize = (nodeTotalCount + blockSize - 1) / blockSize;
		createSVO_PG_device << <gridSize, blockSize, 0, sourceManager->stream >> > (SVONodes_children, SVONodes, nodeCount);
	}
	
	//ѹ������
	if (setting.voxelNum > 128) {
		std::cout << "voxelNum����128������Ҫ����ѹ����Ŀǰ�޷�ʵ�֣�SVOδѹ��" << std::endl;
		return;
	}
	uint32_t blockInfoIndex = 0;
	for (int i = 0; i < SVOs_PG.size(); ++i) {
		uint32_t nodeTotalCount = pow(8, i + 1);
		blockSize = nodeTotalCount > createSVOKernelBlockSize ? createSVOKernelBlockSize : nodeTotalCount;
		gridSize = (nodeTotalCount + blockSize - 1) / blockSize;

		FzbSVONodeData_PG* SVONodes = SVOs_PG[i];
		uint32_t* svoNodeCount = SVONodeCount[i];
		if (nodeTotalCount <= 512) compressSVO_PG_firstStep<false><<<gridSize, blockSize, 0, sourceManager->stream>>>
			(SVONodes, nullptr, nullptr, svoNodeCount, nodeTotalCount);
		else {
			FzbSVONodeBlock* blockInfo = SVONodeBlockInfos[blockInfoIndex];
			FzbSVONodeTempInfo* tempNodeInfo = SVONodeTempInfos[blockInfoIndex++];
			compressSVO_PG_firstStep<true> <<<gridSize, blockSize, 0, sourceManager->stream >>> 
				(SVONodes, blockInfo, tempNodeInfo, svoNodeCount, nodeTotalCount);
			compressSVO_PG_secondStep <<<gridSize, blockSize, 0, sourceManager->stream >>>
				(SVONodes, blockInfo, tempNodeInfo, nodeTotalCount);
		}
	}
}
//----------------------------------------------------------------------------------------------------------------------------------------------------
void FzbSVOCuda_PG::clean() {
	CHECK(hipDestroyExternalMemory(VGBExtMem));
	CHECK(hipFree(VGB));
	for (int i = 0; i < this->SVONodeBlockInfos.size(); ++i) CHECK(hipFree(this->SVONodeBlockInfos[i]));
	for (int i = 0; i < this->SVONodeTempInfos.size(); ++i) CHECK(hipFree(this->SVONodeTempInfos[i]));
	for (int i = 0; i < this->SVONodeCount.size(); ++i) CHECK(hipFree(this->SVONodeCount[i]));
	for (int i = 0; i < this->SVOs_PG.size(); ++i) CHECK(hipFree(this->SVOs_PG[i]));
	CHECK(hipDestroyExternalSemaphore(extSvoSemaphore_PG));
}

void FzbSVOCuda_PG::copyDataToBuffer(std::vector<FzbBuffer>& buffers) {
	if (buffers.size() != SVOs_PG.size()) throw std::runtime_error("SVOBuffer������ƥ��");
	for (int i = 0; i < buffers.size(); ++i) {
		FzbBuffer& SVOBuffer = buffers[i];
		hipExternalMemory_t SVOBufferExtMem = importVulkanMemoryObjectFromNTHandle(SVOBuffer.handle, SVOBuffer.size, false);
		FzbSVONodeData_PG* SVOBuffer_ptr = (FzbSVONodeData_PG*)mapBufferOntoExternalMemory(SVOBufferExtMem, 0, SVOBuffer.size);
		CHECK(hipMemcpy(SVOBuffer_ptr, SVOs_PG[i], SVOBuffer.size, hipMemcpyDeviceToDevice));
		CHECK(hipDestroyExternalMemory(SVOBufferExtMem));
		CHECK(hipFree(SVOBuffer_ptr));
	}
}
