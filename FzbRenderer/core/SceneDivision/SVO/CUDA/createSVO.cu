#include "hip/hip_runtime.h"
#pragma once

#include "./createSVO.cuh"

#ifndef CREATE_SVO_CU
#define CREATE_SVO_CU

//-------------------------------------------------------------------�˺���------------------------------------------------------------------
/*
__global__ void getSVONum(hipTextureObject_t voxelGridMap, uint32_t* voxelNum, FzbSVONode* svoNodeArray, uint32_t svoDepth, FzbVoxelValue* svoVoxelValueArray) {

	//extern __shared__ int subSVONodeNum[];	//�����ڴ�����Ϊ�ȱ�������ͣ����Ƿ����ⲿ����
	glm::uvec3 voxelIndexU3 = glm::uvec3(blockDim.x * blockIdx.x + threadIdx.x, blockDim.y * blockIdx.y + threadIdx.y, blockDim.z * blockIdx.z + threadIdx.z);

	//���ﲻ֪������ȡ�Ƿ��죬֮�����һ��
	uint32_t valueU = tex3D<uint32_t>(voxelGridMap, voxelIndexU3.x, voxelIndexU3.y, voxelIndexU3.z);
	//glm::vec4 value = glm::unpackUnorm4x8(valueU);
	if (valueU <= 0) {
		return;
	}
	uint32_t svoVoxelValueIndex = atomicAdd(voxelNum, 1);
	svoVoxelValueArray[svoVoxelValueIndex].pos_num = valueU;

	atomicAdd(&svoNodeArray[0].voxelNum, 1);
	int fatherNodeIndex = 0;
	int curNodeIndex = 0;
	int detailLevel = gridDim.x * blockDim.x;
	for (int i = 1; i < svoDepth; i++) {
		uint3 index;
		index.x = voxelIndexU3.x % detailLevel;
		index.y = voxelIndexU3.y % detailLevel;
		index.z = voxelIndexU3.z % detailLevel;
		detailLevel /= 2;
		index.x /= detailLevel;
		index.y /= detailLevel;
		index.z /= detailLevel;

		int subNodeIndex = index.y * 2 + index.x + index.z * 4;
		curNodeIndex = subNodeIndex + fatherNodeIndex * 8 + 1;
		atomicAdd(&svoNodeArray[curNodeIndex].voxelNum, 1);

		uint32_t hasSubNode = 1 << subNodeIndex;
		atomicOr(&svoNodeArray[fatherNodeIndex].hasSubNode, hasSubNode);

		fatherNodeIndex = curNodeIndex;
	}
	svoNodeArray[curNodeIndex].subsequentIndex = svoVoxelValueIndex;

}
__global__ void getSVONum_UseShared(hipTextureObject_t voxelGridMap, uint32_t* voxelNum, FzbSVONode* svoNodeArray, uint32_t svoDepth, FzbVoxelValue* svoVoxelValueArray) {

	__shared__ uint32_t svoVoxelValueIndex;
	__shared__ uint32_t voxelValueGroupNum;
	extern __shared__ FzbSVONode subSVONodeSharedArray[];	//�����ڴ�����Ϊ�ȱ�������ͣ����Ƿ����ⲿ���㡣��༸�������м�ڵ�

	uint3 voxelIndexU3 = make_uint3(blockDim.x * blockIdx.x + threadIdx.x, blockDim.y * blockIdx.y + threadIdx.y, blockDim.z * blockIdx.z + threadIdx.z);
	uint32_t threadGroupIndex = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
	uint32_t laneIndex = threadGroupIndex % warpSize;	//һ��warp 32���߳�

	FzbSVONode initNode;
	if (threadGroupIndex == 0) {
		voxelValueGroupNum = 0;
	}
	if (threadGroupIndex < svoDepth + 69) {	//68 = 8 * 8 + 8 - 3
		subSVONodeSharedArray[threadGroupIndex] = initNode;
	}

	uint32_t valueU = tex3D<uint32_t>(voxelGridMap, voxelIndexU3.x, voxelIndexU3.y, voxelIndexU3.z);
	uint32_t hasValue = valueU > 0 ? 1 : 0;
	uint32_t warpVoxelNum = warpReduce(hasValue);
	if (laneIndex == 0)
		atomicAdd(&subSVONodeSharedArray[0].voxelNum, warpVoxelNum);
	__syncthreads();
	uint32_t blockVoxelNum = subSVONodeSharedArray[0].voxelNum;
	if (blockVoxelNum == 0)
		return;
	if (threadGroupIndex == 0)
		svoVoxelValueIndex = atomicAdd(voxelNum, blockVoxelNum);

	int curNodeIndex = 0;
	uint32_t sharedDataIndex = 0;
	int detailLevel = gridDim.x * blockDim.x;
	int subNodeIndex = 0;
	uint32_t sharedDataOffset = svoDepth - 4;
	for (int i = 0; i < svoDepth - 4; i++) {
		if (threadGroupIndex == 0)
			subSVONodeSharedArray[i].nodeIndex = curNodeIndex;

		uint3 index;
		index.x = voxelIndexU3.x & (detailLevel - 1);	//�൱��voxelIndexU3.x % detailLevel
		index.y = voxelIndexU3.y & (detailLevel - 1);
		index.z = voxelIndexU3.z & (detailLevel - 1);
		detailLevel /= 2;
		index.x /= detailLevel;
		index.y /= detailLevel;
		index.z /= detailLevel;

		subNodeIndex = index.y * 2 + index.x + index.z * 4;
		curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;

		if (threadGroupIndex == 0) {
			uint32_t hasSubNode = 1 << subNodeIndex;
			subSVONodeSharedArray[i].hasSubNode = hasSubNode;
		}
	}
	if (threadGroupIndex == 0) {
		subSVONodeSharedArray[sharedDataOffset].nodeIndex = curNodeIndex;
	}
	if (valueU > 0) {
		for (int i = 0; i < 2; i++) {

			uint3 index;
			index.x = voxelIndexU3.x & (detailLevel - 1);
			index.y = voxelIndexU3.y & (detailLevel - 1);
			index.z = voxelIndexU3.z & (detailLevel - 1);
			detailLevel /= 2;
			index.x /= detailLevel;
			index.y /= detailLevel;
			index.z /= detailLevel;

			subNodeIndex = index.y * 2 + index.x + index.z * 4;
			curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;

			uint32_t hasSubNode = 1 << subNodeIndex;
			atomicOr(&subSVONodeSharedArray[sharedDataIndex + sharedDataOffset].hasSubNode, hasSubNode);

			sharedDataIndex = sharedDataIndex * 8 + 1 + subNodeIndex;
			atomicAdd(&subSVONodeSharedArray[sharedDataIndex + sharedDataOffset].voxelNum, 1);
			subSVONodeSharedArray[sharedDataIndex + sharedDataOffset].nodeIndex = curNodeIndex;
		}

		uint3 index;
		index.x = voxelIndexU3.x & (detailLevel - 1);
		index.y = voxelIndexU3.y & (detailLevel - 1);
		index.z = voxelIndexU3.z & (detailLevel - 1);
		detailLevel /= 2;
		index.x /= detailLevel;
		index.y /= detailLevel;
		index.z /= detailLevel;

		subNodeIndex = index.y * 2 + index.x + index.z * 4;
		curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;

		uint32_t hasSubNode = 1 << subNodeIndex;
		atomicOr(&subSVONodeSharedArray[sharedDataIndex + sharedDataOffset].hasSubNode, hasSubNode);
		svoNodeArray[curNodeIndex].voxelNum = 1;
	}
	__syncthreads();

	if (valueU > 0) {
		uint32_t voxelValueGroupIndex = atomicAdd(&voxelValueGroupNum, 1);
		svoVoxelValueArray[svoVoxelValueIndex + voxelValueGroupIndex].pos_num = valueU;
		svoNodeArray[curNodeIndex].subsequentIndex = svoVoxelValueIndex + voxelValueGroupIndex;
	}

	if (threadGroupIndex < svoDepth - 3) {
		atomicAdd(&svoNodeArray[subSVONodeSharedArray[threadGroupIndex].nodeIndex].voxelNum, blockVoxelNum);
		atomicOr(&svoNodeArray[subSVONodeSharedArray[threadGroupIndex].nodeIndex].hasSubNode, subSVONodeSharedArray[threadGroupIndex].hasSubNode);
	}
	if (threadGroupIndex < 8) {
		sharedDataIndex = svoDepth - 3 + threadGroupIndex;
		if (subSVONodeSharedArray[sharedDataIndex].voxelNum > 0) {
			atomicAdd(&svoNodeArray[subSVONodeSharedArray[sharedDataIndex].nodeIndex].voxelNum, subSVONodeSharedArray[sharedDataIndex].voxelNum);
			atomicOr(&svoNodeArray[subSVONodeSharedArray[sharedDataIndex].nodeIndex].hasSubNode, subSVONodeSharedArray[sharedDataIndex].hasSubNode);
		}
	}
	if (threadGroupIndex < 64) {
		sharedDataIndex = svoDepth + 5 + threadGroupIndex;	//5 = 8 - 3
		if (subSVONodeSharedArray[sharedDataIndex].voxelNum > 0) {
			atomicAdd(&svoNodeArray[subSVONodeSharedArray[sharedDataIndex].nodeIndex].voxelNum, subSVONodeSharedArray[sharedDataIndex].voxelNum);
			atomicOr(&svoNodeArray[subSVONodeSharedArray[sharedDataIndex].nodeIndex].hasSubNode, subSVONodeSharedArray[sharedDataIndex].hasSubNode);
		}
	}
}

__global__ void getSVONum_step1(hipTextureObject_t voxelGridMap, uint32_t svoDepth, uint32_t nonLeafNodeNum, uint32_t* voxelNum, FzbSVONode* svoNodeArray, FzbVoxelValue* svoVoxelValueArray) {

	__shared__ uint32_t groupVoxelNum;
	__shared__ uint32_t groupVoxelOffset;

	uint3 voxelIndexU3 = make_uint3(blockDim.x * blockIdx.x + threadIdx.x, blockDim.y * blockIdx.y + threadIdx.y, blockDim.z * blockIdx.z + threadIdx.z);
	uint32_t threadGroupIndex = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
	uint32_t warpIndex = threadGroupIndex / warpSize;	//һ��warp 32���߳�
	uint32_t laneIndex = threadGroupIndex % warpSize;

	if (threadGroupIndex == 0) {
		groupVoxelNum = 0;
		groupVoxelOffset = 0;
	}
	__syncthreads();

	uint32_t valueU = tex3D<uint32_t>(voxelGridMap, voxelIndexU3.x, voxelIndexU3.y, voxelIndexU3.z);
	uint32_t voxelLocalIndex;
	if(valueU > 0)
		voxelLocalIndex = atomicAdd(&groupVoxelNum, 1);
	__syncthreads();
	if (threadGroupIndex == 0)
		groupVoxelOffset = atomicAdd(voxelNum, groupVoxelNum);
	__syncthreads();

	if (valueU > 0) {
		uint32_t voxelIndexU = packUint3(voxelIndexU3);
		svoNodeArray[voxelLocalIndex + groupVoxelOffset + nonLeafNodeNum].shuffleKey = voxelIndexU;	//����Ҷ�ӽڵ��λ�ã���ռ֮ǰ�Ĺ����ڵ��λ��
		svoVoxelValueArray[voxelLocalIndex + groupVoxelOffset].pos_num = valueU;
	}

	if (threadGroupIndex == 0) {
		atomicAdd(&svoNodeArray[0].voxelNum, groupVoxelNum);

		int curNodeIndex = 0;
		int detailLevel = gridDim.x * blockDim.x;
		int subNodeIndex = 0;
		uint32_t shuffleKey = 0;
		for (int i = 1; i < svoDepth - 3; i++) {

			uint3 index = make_uint3(voxelIndexU3.x & (detailLevel - 1), voxelIndexU3.y & (detailLevel - 1), voxelIndexU3.z & (detailLevel - 1));
			detailLevel /= 2;
			index.x /= detailLevel;
			index.y /= detailLevel;
			index.z /= detailLevel;
			subNodeIndex = index.y * 2 + index.x + index.z * 4;
			shuffleKey = (shuffleKey << 3) | subNodeIndex;

			uint32_t hasSubNode = 1 << subNodeIndex;
			if (i < svoDepth - 4)
				atomicOr(&svoNodeArray[curNodeIndex].hasSubNode, hasSubNode);

			curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;
			atomicAdd(&svoNodeArray[curNodeIndex].voxelNum, groupVoxelNum);
			svoNodeArray[curNodeIndex].shuffleKey = shuffleKey;
		}
	}

}
__global__ void getSVONum_step2(uint32_t voxelNum, uint32_t svoDepth, uint32_t nonLeafNodeNum, uint32_t svoSize, FzbSVONode* svoNodeArray) {

	uint32_t localThreadIndex = threadIdx.x;
	uint32_t globalThreadIndex = blockIdx.x * blockDim.x + localThreadIndex;
	if (globalThreadIndex >= voxelNum)
		return;

	uint32_t voxelIndex = globalThreadIndex + nonLeafNodeNum;
	FzbSVONode voxelIndexU3Info = svoNodeArray[voxelIndex];
	uint3 voxelIndexU3 = unpackUint(voxelIndexU3Info.shuffleKey);

	int curNodeIndex = 0;
	int detailLevel = svoSize;
	int subNodeIndex = 0;
	uint32_t shuffleKey = 0;
	for (int i = 0; i < svoDepth - 4; i++) {
		uint3 index = make_uint3(voxelIndexU3.x & (detailLevel - 1), voxelIndexU3.y & (detailLevel - 1), voxelIndexU3.z & (detailLevel - 1));
		detailLevel /= 2;
		index.x /= detailLevel;
		index.y /= detailLevel;
		index.z /= detailLevel;
		subNodeIndex = index.y * 2 + index.x + index.z * 4;
		shuffleKey = (shuffleKey << 3) | subNodeIndex;
		curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;
	}

	for (int i = 0; i < 2; i++) {
		uint3 index = make_uint3(voxelIndexU3.x & (detailLevel - 1), voxelIndexU3.y & (detailLevel - 1), voxelIndexU3.z & (detailLevel - 1));
		detailLevel /= 2;
		index.x /= detailLevel;
		index.y /= detailLevel;
		index.z /= detailLevel;

		subNodeIndex = index.y * 2 + index.x + index.z * 4;
		shuffleKey = (shuffleKey << 3) | subNodeIndex;
		uint32_t hasSubNode = 1 << subNodeIndex;
		atomicOr(&svoNodeArray[curNodeIndex].hasSubNode, hasSubNode);

		curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;

		atomicAdd(&svoNodeArray[curNodeIndex].voxelNum, 1);
		svoNodeArray[curNodeIndex].shuffleKey = shuffleKey;
	}
	uint3 index = make_uint3(voxelIndexU3.x & (detailLevel - 1), voxelIndexU3.y & (detailLevel - 1), voxelIndexU3.z & (detailLevel - 1));
	detailLevel /= 2;
	index.x /= detailLevel;
	index.y /= detailLevel;
	index.z /= detailLevel;
	subNodeIndex = index.y * 2 + index.x + index.z * 4;
	shuffleKey = (shuffleKey << 3) | subNodeIndex;

	uint32_t hasSubNode = 1 << subNodeIndex;
	atomicOr(&svoNodeArray[curNodeIndex].hasSubNode, hasSubNode);

	curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;
	svoNodeArray[curNodeIndex].voxelNum = 1;
	svoNodeArray[curNodeIndex].shuffleKey = shuffleKey;
	svoNodeArray[curNodeIndex].subsequentIndex = globalThreadIndex;
}

__global__ void compressSVO(FzbSVONode* nodeArray, FzbSVONode* nodePool, uint32_t nodeStartIndex, uint32_t subArrayStartIndex, uint32_t* subArrayNum, glm::vec4 fatherNodePos_Size, uint32_t nonLeafNodeNum) {

	int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

	int nodeIndex = nodeStartIndex + threadIndex;
	FzbSVONode node = nodeArray[nodeIndex];
	if (node.voxelNum == 0)
		return;

	float nodeSize = fatherNodePos_Size.w / 2;
	glm::vec4 nodePos_Size = fatherNodePos_Size;
	nodePos_Size.x += nodeSize * (threadIndex % 2);
	nodePos_Size.y += nodeSize * ((threadIndex % 4) / 2);
	nodePos_Size.z += nodeSize * (threadIndex / 4);
	nodePos_Size.w = nodeSize;

	node.nodePos_Size = nodePos_Size;

	int nodePoolIndex = subArrayStartIndex + threadIndex;
	nodePool[nodePoolIndex] = node;

	if (nodeStartIndex >= nonLeafNodeNum)
		return;
	int subArrayIndex = atomicAdd(subArrayNum, 1);
	nodePool[nodePoolIndex].subsequentIndex = subArrayIndex * 8;
	compressSVO << <1, 8 >> > (nodeArray, nodePool, nodeIndex * 8 + 1, subArrayIndex * 8, subArrayNum, nodePos_Size, nonLeafNodeNum);

}
*/
/*
getSVONum_step1�ҵ�������ֵ�����أ�������������svoNodeArray��Ҷ�ڵ��shuffleKey�У�Ȼ��Ը��߳���Ĺ����ڵ㸳ֵ
getSVONum_step2���̶߳�Ӧ����ֵ���أ��ҵ������м�ڵ㲢��ֵ�����յõ����˲�������
compressSVO_Step1����ִ�С�ʹ��ÿһ����ÿ���߳���������
compressSVO_Step2ͬ������ִ�У����߳����������յõ������ѹ�����顣
*/
__global__ void getSVONum_step1(hipTextureObject_t voxelGridMap, uint32_t svoDepth, uint32_t nonLeafNodeNum, uint32_t* voxelNum, FzbSVONode* svoNodeArray, FzbVoxelValue* svoVoxelValueArray) {

	__shared__ uint32_t groupVoxelNum;
	__shared__ uint32_t groupVoxelOffset;

	uint3 voxelIndexU3 = make_uint3(blockDim.x * blockIdx.x + threadIdx.x, blockDim.y * blockIdx.y + threadIdx.y, blockDim.z * blockIdx.z + threadIdx.z);
	uint32_t threadGroupIndex = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
	uint32_t warpIndex = threadGroupIndex / warpSize;	//һ��warp 32���߳�
	uint32_t laneIndex = threadGroupIndex % warpSize;

	if (threadGroupIndex == 0) {
		groupVoxelNum = 0;
		groupVoxelOffset = 0;
	}
	__syncthreads();

	uint32_t valueU = tex3D<uint32_t>(voxelGridMap, voxelIndexU3.x, voxelIndexU3.y, voxelIndexU3.z);
	uint32_t voxelLocalIndex;
	if (valueU > 0)
		voxelLocalIndex = atomicAdd(&groupVoxelNum, 1);
	__syncthreads();
	if (threadGroupIndex == 0)
		groupVoxelOffset = atomicAdd(voxelNum, groupVoxelNum);
	__syncthreads();

	if (valueU > 0) {
		uint32_t voxelIndexU = packUint3(voxelIndexU3);
		svoNodeArray[voxelLocalIndex + groupVoxelOffset + nonLeafNodeNum].shuffleKey = voxelIndexU;	//����Ҷ�ӽڵ��λ�ã���ռ֮ǰ�Ĺ����ڵ��λ��
		svoVoxelValueArray[voxelLocalIndex + groupVoxelOffset].pos_num = valueU;
	}

	if (threadGroupIndex == 0 && groupVoxelNum > 0) {
		atomicAdd(&svoNodeArray[0].voxelNum, groupVoxelNum);

		int curNodeIndex = 0;
		int detailLevel = gridDim.x * blockDim.x;
		int subNodeIndex = 0;
		for (int i = 1; i < svoDepth - 3; i++) {

			uint3 index = make_uint3(voxelIndexU3.x & (detailLevel - 1), voxelIndexU3.y & (detailLevel - 1), voxelIndexU3.z & (detailLevel - 1));
			detailLevel /= 2;
			index.x /= detailLevel;
			index.y /= detailLevel;
			index.z /= detailLevel;
			subNodeIndex = index.y * 2 + index.x + index.z * 4;

			uint32_t hasSubNode = 1 << subNodeIndex;
			if (i < svoDepth - 4)
				atomicOr(&svoNodeArray[curNodeIndex].hasSubNode, hasSubNode);

			curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;
			atomicAdd(&svoNodeArray[curNodeIndex].voxelNum, groupVoxelNum);
		}
	}

}
__global__ void getSVONum_step2(uint32_t voxelNum, uint32_t svoDepth, uint32_t nonLeafNodeNum, uint32_t svoSize, FzbSVONode* svoNodeArray) {

	uint32_t localThreadIndex = threadIdx.x;
	uint32_t globalThreadIndex = blockIdx.x * blockDim.x + localThreadIndex;
	if (globalThreadIndex >= voxelNum)
		return;

	uint32_t voxelIndex = globalThreadIndex + nonLeafNodeNum;
	FzbSVONode voxelIndexU3Info = svoNodeArray[voxelIndex];
	uint3 voxelIndexU3 = unpackUint(voxelIndexU3Info.shuffleKey);

	int curNodeIndex = 0;
	int detailLevel = svoSize;
	int subNodeIndex = 0;
	for (int i = 0; i < svoDepth - 4; i++) {
		uint3 index = make_uint3(voxelIndexU3.x & (detailLevel - 1), voxelIndexU3.y & (detailLevel - 1), voxelIndexU3.z & (detailLevel - 1));
		detailLevel /= 2;
		index.x /= detailLevel;
		index.y /= detailLevel;
		index.z /= detailLevel;
		subNodeIndex = index.y * 2 + index.x + index.z * 4;
		curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;
	}

	for (int i = 0; i < 2; i++) {
		uint3 index = make_uint3(voxelIndexU3.x & (detailLevel - 1), voxelIndexU3.y & (detailLevel - 1), voxelIndexU3.z & (detailLevel - 1));
		detailLevel /= 2;
		index.x /= detailLevel;
		index.y /= detailLevel;
		index.z /= detailLevel;

		subNodeIndex = index.y * 2 + index.x + index.z * 4;
		uint32_t hasSubNode = 1 << subNodeIndex;
		atomicOr(&svoNodeArray[curNodeIndex].hasSubNode, hasSubNode);

		curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;

		atomicAdd(&svoNodeArray[curNodeIndex].voxelNum, 1);
	}
	uint3 index = make_uint3(voxelIndexU3.x & (detailLevel - 1), voxelIndexU3.y & (detailLevel - 1), voxelIndexU3.z & (detailLevel - 1));
	detailLevel /= 2;
	index.x /= detailLevel;
	index.y /= detailLevel;
	index.z /= detailLevel;
	subNodeIndex = index.y * 2 + index.x + index.z * 4;

	uint32_t hasSubNode = 1 << subNodeIndex;
	atomicOr(&svoNodeArray[curNodeIndex].hasSubNode, hasSubNode);

	curNodeIndex = subNodeIndex + curNodeIndex * 8 + 1;
	svoNodeArray[curNodeIndex].voxelNum = 1;
	svoNodeArray[curNodeIndex].label = globalThreadIndex;
}

template <int type>
__global__ void compressSVO_Step1(FzbSVONode* nodeArray, FzbSVONode* tempNodePool, FzbSVONode* nodePool, uint32_t* subArrayNum, FzbNodePoolBlock* threadBlockInfos, uint32_t nodeStartIndex,uint32_t svoDepth) {
	__shared__ uint64_t blockHasValue;	//ÿ���˸��߳�һ��Ԫ��, 8x64 = 512��һ���߳������512���߳�
	__shared__ uint32_t blockIndexOfGroup;
	__shared__ uint32_t nodeNum;
	__shared__ uint32_t blockNodeNum[64];

	uint32_t threadLocalIndex = threadIdx.x;
	uint32_t threadGlobalIndex = blockDim.x * blockIdx.x + threadLocalIndex;
	uint32_t blockIndexInGroup = threadLocalIndex / 8;
	uint32_t threadOffsetInBlock = threadLocalIndex & 7;
	if (threadLocalIndex == 0) {
		blockHasValue = 0;
		blockIndexOfGroup = 0;
		nodeNum = 0;
	}
	if (threadGlobalIndex < 64)
		blockNodeNum[threadGlobalIndex] = 0;
	__syncthreads();

	uint32_t nodeIndex = threadGlobalIndex + nodeStartIndex;
	FzbSVONode node = nodeArray[nodeIndex];
	FzbSVONode fatherNode = nodeArray[nodeIndex / 8];
	if (node.voxelNum > 0) {	//ǰ���64��512����Ҫ
		if (type == 2) {
			atomicAdd(&nodeNum, 1);
		}
		atomicAdd(&blockNodeNum[blockIndexInGroup], 1);
	}
	if (fatherNode.voxelNum > 0) {
		if (threadOffsetInBlock == 0)
			atomicOr(&blockHasValue, uint64_t(1) << blockIndexInGroup);
	}
	__syncthreads();

	if (threadLocalIndex == 0) {
		uint32_t blockNum = __popcll(blockHasValue);
		blockIndexOfGroup = atomicAdd(subArrayNum, blockNum);	//�߳�����ȫ�ֿ��������ʼ����
		if (type == 2)	//ǰ���64��512����Ҫ
			threadBlockInfos[blockIdx.x] = FzbNodePoolBlock(blockIndexOfGroup, blockNum, nodeNum);
	}
	__syncthreads();

	uint32_t blockIndex = __popcll(blockHasValue & ((uint64_t(1) << blockIndexInGroup) - 1)) + blockIndexOfGroup;	//�߳�����block��ȫ���е�����
	if (node.voxelNum > 0) {
		uint32_t label = 0;
		for (int i = 0; i < blockIndexInGroup; i++) {
			label += blockNodeNum[i];
		}
		node.label = __popcll(fatherNode.hasSubNode & ((1u << threadOffsetInBlock) - 1)) + label + 1;
		//uint32_t blockIndexInWarp = threadLocalIndex & 3;
		//uint32_t offset = 8 * blockIndexInWarp;
		//node.label = __popc(((__ballot_sync(__activemask(), 1) & (0xFF << offset)) >> offset) & ((1u << threadOffsetInBlock) - 1)) + label + 1;
		if (type == 2) {
			tempNodePool[1 + blockIndex * 8 + threadOffsetInBlock] = node;
		}
		else {
			nodePool[1 + blockIndex * 8 + threadOffsetInBlock] = node;
		}
	}
	if (blockNodeNum[blockIndexInGroup] > 0) {
		if (type == 2) {
			tempNodePool[1 + blockIndex * 8 + threadOffsetInBlock].shuffleKey = threadGlobalIndex | (svoDepth << 28);
		}
		else {
			nodePool[1 + blockIndex * 8 + threadOffsetInBlock].shuffleKey = threadGlobalIndex | (svoDepth << 28);
		}
	}

	//�����ڵ�͵�һ��ڵ����
	if (type == 0) {
		FzbSVONode rootNode = nodeArray[0];
		rootNode.label = 1;
		if (threadGlobalIndex == 0)
			nodePool[0] = rootNode;
		if (threadGlobalIndex < 8) {
			FzbSVONode node = nodeArray[threadGlobalIndex + 1];
			node.shuffleKey = threadGlobalIndex | (uint32_t(1) << 28);
			node.label = __popc(rootNode.hasSubNode & ((1u << threadGlobalIndex) - 1)) + 1;
			nodePool[threadGlobalIndex + 1] = node;
		}

	}
}

__global__ void compressSVO_Step2(FzbNodePoolBlock* threadBlockInfos, FzbSVONode* tempNodePool, FzbSVONode* nodePool) {
	__shared__ FzbNodePoolBlock threadBlockInfo;
	__shared__ uint32_t firstBlockIndex;

	uint32_t threadLocalIndex = threadIdx.x;
	uint32_t threadGlobalIndex = blockDim.x * blockIdx.x + threadLocalIndex;

	if (threadLocalIndex == 0) {
		threadBlockInfo = threadBlockInfos[blockIdx.x];
		firstBlockIndex = threadBlockInfos[0].startIndex;
	}
	__syncthreads();

	if (threadLocalIndex >= threadBlockInfo.blockNum * 8)
		return;

	uint32_t nodeIndex = threadBlockInfo.startIndex * 8 + threadLocalIndex + 1;
	FzbSVONode node = tempNodePool[nodeIndex];
	uint32_t newNodeIndex = firstBlockIndex * 8 + threadLocalIndex + 1;
	uint32_t label = 0;
	for (int i = 0; i < blockIdx.x; i++) {
		FzbNodePoolBlock blockInfo = threadBlockInfos[i];
		label += blockInfo.nodeNum;
		newNodeIndex += blockInfo.blockNum * 8;
	}
	node.label += label;
	nodePool[newNodeIndex] = node;
}

//-------------------------------------------------------------------------------------------------------------------------
/*
void CUDART_CB cleanTempData(hipStream_t stream, hipError_t status, void* userData) {

	SVOCuda* svoCuda = (SVOCuda*)userData;

	CHECK(hipDestroyExternalSemaphore(svoCuda->extVgmSemaphore));
	CHECK(hipDestroyExternalSemaphore(svoCuda->extSvoSemaphore));
	CHECK(hipDestroyTextureObject(svoCuda->vgm));
	CHECK(hipFreeMipmappedArray(svoCuda->vgmMipmap));
	CHECK(hipDestroyExternalMemory(svoCuda->vgmExtMem));
	CHECK(hipDestroyExternalMemory(svoCuda->nodePoolExtMem));
	CHECK(hipDestroyExternalMemory(svoCuda->voxelValueArrayExtMem));

	CHECK(hipHostFree(svoCuda->voxelNum));
	CHECK(hipHostFree(svoCuda->nodeArrayNum));
	CHECK(hipHostFree(svoCuda->subArrayNum_host));

	CHECK(hipFreeAsync(svoCuda->voxelNum_p, stream));
	CHECK(hipFreeAsync(svoCuda->svoNodeArray, stream));
	CHECK(hipFreeAsync(svoCuda->subArrayNum, stream));

	CHECK(hipFreeAsync(svoCuda->nodePool, stream));
	CHECK(hipFreeAsync(svoCuda->svoVoxelValueArray, stream));

	CHECK(hipStreamDestroy(svoCuda->stream));

}
*/
void SVOCuda::createSVOCuda(VkPhysicalDevice vkPhysicalDevice, FzbImage& voxelGridMap, HANDLE vgmSemaphoreHandle, HANDLE svoSemaphoreHandle, bool isPresent) {

	double start = cpuSecond();

	unsigned long long size = voxelGridMap.width * voxelGridMap.height * voxelGridMap.depth * sizeof(uint32_t);
	fromVulkanImageToCudaTexture(vkPhysicalDevice, voxelGridMap, voxelGridMap.handle, size, false, vgmExtMem, vgmMipmap, vgm);

	extVgmSemaphore = importVulkanSemaphoreObjectFromNTHandle(vgmSemaphoreHandle);
	extSvoSemaphore = importVulkanSemaphoreObjectFromNTHandle(svoSemaphoreHandle);

	dim3 gridSize(voxelGridMap.width / 8, voxelGridMap.height / 8, voxelGridMap.depth / 8);
	dim3 blockSize(8, 8, 8);
	//���SVO�����
	uint32_t svoDepth = 1;
	uint32_t vgmSize = voxelGridMap.width;
	while (vgmSize > 1) {
		svoDepth++;
		vgmSize >>= 1;
	}
	uint32_t maxNodeNum = uint32_t((pow(8, svoDepth) - 1) / 7);	//���˲����������ڵ���
	uint32_t nonLeafNodeNum = uint32_t((pow(8, svoDepth - 1) - 1) / 7);

	CHECK(hipStreamCreate(&stream));
	uint32_t* voxelNum_p;
	CHECK(hipMalloc((void**)&voxelNum_p, sizeof(uint32_t)));	//��ʹ�ù̶��ڴ棬���޷�ʹ��ԭ������
	CHECK(hipMemset(voxelNum_p, 0, sizeof(uint32_t)));

	//����һ�����˲������飬�����ŷ�
	FzbSVONode* svoNodeArray;
	CHECK(hipMalloc((void**)&svoNodeArray, sizeof(FzbSVONode) * maxNodeNum));
	CHECK(hipMemset(svoNodeArray, 0, sizeof(FzbSVONode) * maxNodeNum));
	//����һ����ֵ�������ݵ�����
	CHECK(hipMalloc((void**)&svoVoxelValueArray, sizeof(FzbVoxelValue) * voxelGridMap.width * voxelGridMap.height * voxelGridMap.depth));
	CHECK(hipMemset(svoVoxelValueArray, 0, sizeof(FzbVoxelValue) * voxelGridMap.width * voxelGridMap.height * voxelGridMap.depth));

	uint32_t subArrayNum_host = 1;
	uint32_t* subArrayNum;
	CHECK(hipMalloc((void**)&subArrayNum, sizeof(uint32_t)));
	CHECK(hipMemcpy(subArrayNum, &subArrayNum_host, sizeof(uint32_t), hipMemcpyHostToDevice));

	FzbSVONode* tempNodePool;
	CHECK(hipMalloc((void**)&tempNodePool, sizeof(FzbSVONode) * maxNodeNum));
	CHECK(hipMemset(tempNodePool, 0, sizeof(FzbSVONode) * maxNodeNum));

	CHECK(hipMalloc((void**)&nodePool, sizeof(FzbSVONode) * maxNodeNum));
	CHECK(hipMemset(nodePool, 0, sizeof(FzbSVONode) * maxNodeNum));

	FzbNodePoolBlock** threadBlockInfos = (FzbNodePoolBlock**)malloc(sizeof(FzbNodePoolBlock*) * (svoDepth - 4));
	for (int i = 0; i < svoDepth - 4; i++) {
		CHECK(hipMalloc((void**)&threadBlockInfos[i], sizeof(FzbNodePoolBlock) * pow(8, i + 4) / 512));
	}
	waitExternalSemaphore(extVgmSemaphore, stream);
	
	getSVONum_step1 << < gridSize, blockSize, 0, stream >> > (vgm, svoDepth, nonLeafNodeNum, voxelNum_p, svoNodeArray, svoVoxelValueArray);
	//����ѹ�����������������
	CHECK(hipMemcpy(&voxelNum, voxelNum_p, sizeof(uint32_t), hipMemcpyDeviceToHost));
	uint32_t blockNum = std::ceil((float)voxelNum / 512);
	getSVONum_step2 << < blockNum, 512, 0, stream >> > (voxelNum, svoDepth, nonLeafNodeNum, voxelGridMap.width, svoNodeArray);

	for (int i = 0; i < svoDepth - 2; i++) {
		uint32_t nodeStartIndex = uint32_t((pow(8, i + 2) - 1) / 7);
		if (i == 0) {
			compressSVO_Step1<0> << <1, 64 >> > (svoNodeArray, tempNodePool, nodePool, subArrayNum, nullptr, nodeStartIndex, 2);
		}
		else if (i == 1) {
			compressSVO_Step1<1> << <1, 512 >> > (svoNodeArray, tempNodePool, nodePool, subArrayNum, nullptr, nodeStartIndex, 3);
		}
		else {
			uint32_t gridSize = pow(8, i + 2) / 512;
			compressSVO_Step1<2> << <gridSize, 512 >> > (svoNodeArray, tempNodePool, nodePool, subArrayNum, threadBlockInfos[i - 2], nodeStartIndex, i+2);
			compressSVO_Step2 << <gridSize, 1024 >> > (threadBlockInfos[i - 2], tempNodePool, nodePool);
		}
	}
	CHECK(hipMemcpy(&nodeBlockNum, subArrayNum, sizeof(uint32_t), hipMemcpyDeviceToHost));

	CHECK(hipDestroyExternalSemaphore(extVgmSemaphore));
	CHECK(hipDestroyExternalSemaphore(extSvoSemaphore));
	CHECK(hipDestroyTextureObject(vgm));
	CHECK(hipFreeMipmappedArray(vgmMipmap));
	CHECK(hipDestroyExternalMemory(vgmExtMem));

	CHECK(hipFree(voxelNum_p));
	CHECK(hipFree(svoNodeArray));
	CHECK(hipFree(subArrayNum));
	CHECK(hipFree(tempNodePool));

	std::cout << cpuSecond() - start << std::endl;

}

void SVOCuda::getSVOCuda(VkPhysicalDevice vkPhysicalDevice, HANDLE nodePoolHandle, HANDLE voxelValueArrayHandle) {
	//���ж��Ƿ���ͬһ�������豸
	if (getCudaDeviceForVulkanPhysicalDevice(vkPhysicalDevice) == hipInvalidDeviceId) {
		throw std::runtime_error("CUDA��Vulkan�õĲ���ͬһ��GPU������");
	}

	nodePoolExtMem = importVulkanMemoryObjectFromNTHandle(nodePoolHandle, sizeof(FzbSVONode) * (8 * nodeBlockNum + 1), false);
	FzbSVONode* vkNodePool = (FzbSVONode*)mapBufferOntoExternalMemory(nodePoolExtMem, 0, sizeof(FzbSVONode) * (8 * nodeBlockNum + 1));
	CHECK(hipMemcpy(vkNodePool, this->nodePool, sizeof(FzbSVONode) * (8 * nodeBlockNum + 1), hipMemcpyDeviceToDevice));

	voxelValueArrayExtMem = importVulkanMemoryObjectFromNTHandle(voxelValueArrayHandle, sizeof(FzbVoxelValue) * voxelNum, false);
	FzbVoxelValue* vkVoxelValueArray = (FzbVoxelValue*)mapBufferOntoExternalMemory(voxelValueArrayExtMem, 0, sizeof(FzbVoxelValue) * voxelNum);
	CHECK(hipMemcpy(vkVoxelValueArray, this->svoVoxelValueArray, sizeof(FzbVoxelValue) * voxelNum, hipMemcpyDeviceToDevice));

	signalExternalSemaphore(extSvoSemaphore, stream);

	//CHECK(hipStreamAddCallback(stream, cleanTempData, this, 0));
	CHECK(hipDestroyExternalMemory(nodePoolExtMem));
	CHECK(hipDestroyExternalMemory(voxelValueArrayExtMem));

	CHECK(hipFree(nodePool));
	CHECK(hipFree(svoVoxelValueArray));

	CHECK(hipStreamDestroy(stream));
}

void SVOCuda::clean() {

}

#endif