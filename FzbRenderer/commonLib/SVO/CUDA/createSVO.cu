#include "hip/hip_runtime.h"
#pragma once

//#include "./createSVO.cuh"
//#include "../../CUDA/vulkanCudaInterop.cuh"
#include "./createSVO.cuh"

#ifndef CREATE_SVO_CU
#define CREATE_SVO_CU

//-------------------------------------------------------------------------------------------------------------------------
struct FzbSVOCudaVariable {
	hipExternalMemory_t extMem;
	hipMipmappedArray_t mipmap;
	hipSurfaceObject_t surfObj = 0;
	hipExternalSemaphore_t extVgmSemaphore;
	hipExternalSemaphore_t extSvoSemaphore;

	FzbSVOCudaVariable() {
		extMem = nullptr;
		mipmap = nullptr;
		surfObj = 0;
		extVgmSemaphore = nullptr;
		extSvoSemaphore = nullptr;
	};
};


__device__ float4 unpackUnorm4x8(uint32_t valueU) {
	float4 value;
	value.x = ((float(valueU & 0xFF)) / 255);
	value.y = ((float((valueU >> 8) & 0xFF)) / 255.0f);
	value.z = ((float((valueU >> 16) & 0xFF)) / 255.0f);
	value.w = ((float((valueU >> 24) & 0xFF)) / 255.0f);
	return value;
}

__device__ uint32_t packUnorm4x8(float4 value) {
	return static_cast<unsigned int>(value.x * 255.0f) | (static_cast<unsigned int>(value.y * 255.0f) << 8)
		| (static_cast<unsigned int>(value.z * 255.0f) << 16) | (static_cast<unsigned int>(value.y * 255.0f) << 24);
}

__global__ void test(hipSurfaceObject_t voxelGridMap) {
	int voxelIndex_x = blockDim.x * blockIdx.x + threadIdx.x;
	int voxelIndex_y = blockDim.y * blockIdx.y + threadIdx.y;
	int voxelIndex_z = blockDim.z * blockIdx.z + threadIdx.z;

	uint32_t valueU = surf3Dread<uint32_t>(voxelGridMap, voxelIndex_x * sizeof(uint32_t), voxelIndex_y, voxelIndex_z, hipBoundaryModeTrap);
	float4 value = unpackUnorm4x8(valueU);
	if (value.w > 0) {
		surf3Dwrite(uint32_t((1 << 32) - 1), voxelGridMap, voxelIndex_x * sizeof(uint32_t), voxelIndex_y, voxelIndex_z);
	}

}

/*
����˺����У����ǿ���64x64x64���̣߳�ÿ��8x8x8���̡߳�
ÿ���߳̿���585��С�Ĺ����ڴ棬�����ڴ��ÿ��Ԫ�ش�������ÿ���ڵ����������������
ͬʱ���������˲����Ľڵ�������
*/
__global__ void getSVONum(hipSurfaceObject_t voxelGridMap, int* svoNodeNum, int* svoNode, int svoDepth_group) {

	extern __shared__ int subSVONodeNum[];	//�����ڴ�����Ϊ�ȱ�������ͣ����Ƿ����ⲿ����
	uint3 voxelIndex;
	voxelIndex.x = blockDim.x * blockIdx.x + threadIdx.x;
	voxelIndex.y = blockDim.y * blockIdx.y + threadIdx.y;
	voxelIndex.z = blockDim.z * blockIdx.z + threadIdx.z;

	//���ﲻ֪������ȡ�Ƿ��죬֮�����һ��
	uint32_t valueU = surf3Dread<uint32_t>(voxelGridMap, voxelIndex.x * sizeof(uint32_t), voxelIndex.y, voxelIndex.z, hipBoundaryModeTrap);
	float4 value = unpackUnorm4x8(valueU);
	if (value.w <= 0) {
		return;
	}
	
	atomicAdd(svoNodeNum, 1);
	atomicAdd(svoNode[0], 1);
	for (int i = 1; i < svoDepth_group; i++) {
		int nodeIndex = gridDim.x
	}

}

//-------------------------------------------------------------------------------------------------------------------------
void createSVOCuda(VkPhysicalDevice vkPhysicalDevice, MyImage& voxelGridMap, HANDLE vgmSemaphoreHandle, HANDLE svoSemaphoreHandle, FzbSVOCudaVariable*& fzbSVOCudaVar) {

	unsigned long long size = voxelGridMap.width * voxelGridMap.height * voxelGridMap.depth * sizeof(uint32_t);
	fzbSVOCudaVar = new FzbSVOCudaVariable();
	fromVulkanImageToCudaSurface(vkPhysicalDevice, voxelGridMap, voxelGridMap.handle, size, true, fzbSVOCudaVar->extMem, fzbSVOCudaVar->mipmap, fzbSVOCudaVar->surfObj);

	fzbSVOCudaVar->extVgmSemaphore = importVulkanSemaphoreObjectFromNTHandle(vgmSemaphoreHandle);
	fzbSVOCudaVar->extSvoSemaphore = importVulkanSemaphoreObjectFromNTHandle(svoSemaphoreHandle);

	hipStream_t stream;
	CHECK(hipStreamCreate(&stream));
	waitExternalSemaphore(fzbSVOCudaVar->extVgmSemaphore, stream);

	dim3 gridSize(voxelGridMap.width / 8, voxelGridMap.height / 8, voxelGridMap.depth / 8);
	dim3 blockSize(8, 8, 8);
	int svoDepth = 0;
	int vgmSize = voxelGridMap.width;
	while (vgmSize > 1) {
		svoDepth++;
		vgmSize >> 1;
	}
	int svoDepth_group = svoDepth - 3;	//ÿһ��ʵ������3��������������������ʣ��Ĳ���

	CHECK(hipStreamSynchronize(stream));
	signalExternalSemaphore(fzbSVOCudaVar->extSvoSemaphore, stream);

}

void cleanSVOCuda(FzbSVOCudaVariable* fzbSVOCudaVar) {
	CHECK(hipDestroyExternalSemaphore(fzbSVOCudaVar->extVgmSemaphore));
	CHECK(hipDestroyExternalSemaphore(fzbSVOCudaVar->extSvoSemaphore));

	CHECK(hipDestroyTextureObject(fzbSVOCudaVar->surfObj));
	CHECK(hipFreeMipmappedArray(fzbSVOCudaVar->mipmap));
	CHECK(hipDestroyExternalMemory(fzbSVOCudaVar->extMem));
}

#endif